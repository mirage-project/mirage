#include "hip/hip_runtime.h"
/* Copyright 2025 CMU
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "mirage/runtime/runtime.h"

namespace mirage {
namespace runtime {

__global__ void init_kernel(RuntimeConfig config) {
  assert(gridDim.x == 1);
  assert(gridDim.y == 1);
  assert(gridDim.z == 1);
  // Only a single thread that initializes everything
  if (threadIdx.x == 0) {
    for (int i = 0; i < config.num_workers; i++) {
      config.worker_queue_last_task_id[i] = 0;
    }
    for (int i = 0; i < config.num_schedulers; i++) {
      config.sched_queue_last_event_id[i] = 0;
    }
  }
}

__device__ int prepare_next_batch(RuntimeConfig config) {
  int batch_size = 0;
}

__device__ void terminate_workers_and_schedulers(RuntimeConfig config) {
  // Send event 0 to all workers
  // Task ID 0 is the termination task
  for (int i = 0; i < config.num_workers; i++) {
    size_t last_task_id = atomicAdd(&config.worker_queue_last_task_id[i], 1);
    config.worker_queues[i][last_task_id % config.per_worker_queue_len] = 0;
  }
  // Event ID 0 is the termination event
  for (int i = 0; i < config.num_schedulers; i++) {
    size_t last_event_id = atomicAdd(&config.sched_queue_last_event_id[i], 1);
    config.sched_queues[i][last_event_id % config.per_sched_queue_len] = 0;
  }
}

__device__ __forceinline__ bool is_termination_event(size_t event_loc,
                                                     EventDesc e) {
  return (event_loc == 0);
}

__global__ void persistent_kernel(RuntimeConfig config) {
  __shared__ TaskId cur_task_loc;
  assert(gridDim.y == 1);
  assert(gridDim.z == 1);
  // Each worker SM serves a single worker
  // Each scheduelr SM serves four schedulers
  assert(config.num_schedulers % 4 == 0);
  assert(gridDim.x == config.num_workers + config.num_schedulers / 4);
  if (blockIdx.x < config.num_workers) {
    int worker_id = blockIdx.x;
    size_t cur_task_id = 0;
    TaskId *task_queue = config.worker_queues[worker_id];
    while (true) {
      // fetch next task from task queue
      if (threadIdx.x == 0) {
        size_t last_task_id = cur_task_id;
        while (cur_task_id == last_task_id) {
          last_task_id = config.worker_queue_last_task_id[worker_id];
        }
        assert(cur_task_id + config.per_worker_queue_len > last_task_id);
        cur_task_loc = task_queue[cur_task_id % config.per_worker_queue_len];
      }
      __syncthreads();
      TaskDesc task_desc = config.all_tasks[cur_task_loc];
      // Successfully fetched a new task
      switch (task_desc.task_type) {
        case TASK_TERMINATE: {
          return;
          break;
        }
        case TASK_RMS_NORM_LINEAR: {
          break;
        }
        default: {
          assert(false && "Unimplemented task");
        }
      }
      __syncthreads();
      // Trigger event
      if (threadIdx.x == 0) {
        EventId event_id = task_desc.trigger_event;
        int count = atomicSub(&config.all_event_counters[event_id], 1);
        if (count == 1) {
          // The event has been triggered enough times
          // Refresh the event counter
          EventDesc event_desc = config.all_events[event_id];
          atomicAdd(&config.all_event_counters[event_id],
                    event_desc.num_triggers);
          // Add the event to the schedule_queue
          int sched_id = event_id % config.num_schedulers;
          size_t last_event_id =
              atomicAdd(&config.sched_queue_last_event_id[sched_id], 1);
          config.sched_queues[sched_id][last_event_id %
                                        config.per_sched_queue_len] = event_id;
        }
      }
      cur_task_id += 1;
    }
  } else {
    // CANNOT use syncthreads on the scheduler side
    int warp_id = threadIdx.x / 32;
    int thread_id = threadIdx.x % 32;
    // assert that we have at least four warps per thread block
    assert(blockDim.x >= 128);
    if (warp_id < 4 && thread_id == 0) {
      int sched_id = (blockIdx.x - config.num_workers) * 4 + warp_id;
      EventId *sched_queue = config.sched_queues[sched_id];
      size_t cur_event_id = 0, last_event_id = 0;
      int next_worker = sched_id * (config.num_workers / config.num_schedulers);
      while (true) {
        while (cur_event_id == last_event_id) {
          last_event_id = config.sched_queue_last_event_id[sched_id];
        }
        // Make sure the schedule queue is not overflow
        assert(cur_event_id + config.per_sched_queue_len > last_event_id);
        // Launch new tasks
        EventId event_id =
            sched_queue[cur_event_id % config.per_sched_queue_len];
        EventDesc e = config.all_events[event_id];
        if (is_termination_event(event_id, e)) {
          return;
        }
        for (TaskId i = e.first_task_id; i < e.last_task_id; i++) {
          size_t last_task_id =
              atomicAdd(&(config.worker_queue_last_task_id[next_worker]), 1);
          config.worker_queues[next_worker]
                              [last_task_id % config.per_worker_queue_len] = i;
          next_worker = (next_worker + 1) % config.num_workers;
        }
        if (e.first_task_id == e.last_task_id) {
          // Terminate all schedulers & workers
          // config.all_tasks[0] and config.all_events[0]
          // are reserved to
          terminate_workers_and_schedulers(config);
          return;
        }
        cur_event_id += 1;
      }
    }
  }
}

void Runtime::launch_persistent_kernel(int num_workers, int num_schedulers) {
  RuntimeConfig config;
  config.num_workers = num_workers;
  config.num_schedulers = num_schedulers;
  config.num_graphs = num_graphs;
  config.total_num_tasks = all_tasks.size();
  config.total_num_events = all_events.size();
  config.per_worker_queue_len = 1024;
  config.per_sched_queue_len = 1024;
  hipMalloc(&config.worker_queue_last_task_id,
             config.num_workers * sizeof(unsigned long long int));
  hipMalloc(&config.sched_queue_last_event_id,
             config.num_schedulers * sizeof(unsigned long long int));
  hipMalloc(&config.all_event_counters, config.total_num_events * sizeof(int));
  // Initialize all tasks
  hipMalloc(&config.all_tasks, config.total_num_tasks * sizeof(TaskDesc));
  hipMemcpy(config.all_tasks,
             all_tasks.data(),
             config.total_num_tasks * sizeof(TaskDesc),
             hipMemcpyHostToDevice);
  // Initialize all events
  hipMalloc(&config.all_events, config.total_num_events * sizeof(EventDesc));
  hipMemcpy(config.all_events,
             all_events.data(),
             config.total_num_events * sizeof(EventDesc),
             hipMemcpyHostToDevice);

  // Launch init kernel
  init_kernel<<<dim3(1, 1, 1), dim3(128, 1, 1)>>>(config);

  // Launcher persistent kernel
  persistent_kernel<<<dim3(108, 1, 1), dim3(128, 1, 1)>>>(config);
}

}; // namespace runtime
}; // namespace mirage
