#include "hip/hip_runtime.h"
/* Copyright 2023-2024 CMU
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cutlass/fast_math.h"
#include "mirage/kernel/device_memory_manager.h"
#include "mirage/kernel/graph.h"
#include "mirage/kernel/rms_norm.h"
#include "mirage/utils/cuda_helper.h"
#include "mirage/utils/hash_utils.h"
#include <cassert>

namespace mirage {
namespace kernel {

bool KNRMSNormOp::profile(ProfileResult &result) {
  // TODO: add profile results
  return true;
}

__global__ void compute_rms_norm_fingerprint(FPType *input_ptr,
                                             FPType *output_ptr,
                                             FPType *div_p_lookup_table,
                                             FPType *div_q_lookup_table,
                                             FPType *sqrt_p_lookup_table,
                                             FPType *sqrt_q_lookup_table,
                                             int num_samples,
                                             int norm_size) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < num_samples) {
    uint32_t square_sum = 0;
    for (int k = 0; k < norm_size; k++) {
      uint32_t x = input_ptr[i * norm_size + k] % FP_PQ;
      x = (x * x) % FP_PQ;
      square_sum = (square_sum + x) % FP_PQ;
    }
    // Compute rooted mean square
    uint32_t rms = 0;
    {
      uint32_t x = square_sum;
      uint32_t n = norm_size;
      // Compute z = x / n
      uint32_t z =
          (x % FP_P) * div_p_lookup_table[n % FP_P] * FP_Q_MUL_P_MOD_1 +
          (x % FP_Q) * div_q_lookup_table[n % FP_Q] * FP_P_MUL_Q_MOD_1;
      // Perform sqrt for root-mean-square
      rms = sqrt_p_lookup_table[z % FP_P] * FP_Q_MUL_P_MOD_1 +
            sqrt_q_lookup_table[z % FP_Q] * FP_P_MUL_Q_MOD_1;
    }
    for (int k = 0; k < norm_size; k++) {
      uint32_t x = input_ptr[i * norm_size + k] % FP_PQ;
      // Compute x / rms
      uint32_t z =
          (x % FP_P) * div_p_lookup_table[rms % FP_P] * FP_Q_MUL_P_MOD_1 +
          (x % FP_Q) * div_q_lookup_table[rms % FP_Q] * FP_P_MUL_Q_MOD_1;
      output_ptr[i * norm_size + k] = z % FP_PQ;
    }
  }
}

bool KNRMSNormOp::fingerprint(void) {
  // assert a 1-D GPU mesh
  assert(kgraph->gpu_dim.y == 1);
  assert(kgraph->gpu_dim.z == 1);
  int num_samples = output_tensors[0].num_elements() / normalized_size;
  int const num_threads_per_blk = 128;
  int num_blocks =
      (num_samples + num_threads_per_blk - 1) / num_threads_per_blk;
  mirage::kernel::DeviceMemoryManager *dmm =
      mirage::kernel::DeviceMemoryManager::get_instance();
  // Use GPU 0 for computing fingerprint
  checkCUDA(hipSetDevice(0));
  for (int gpu_id = 0; gpu_id < kgraph->gpu_dim.x; gpu_id++) {
    mirage::type::FPType *input_fp_ptr =
        reinterpret_cast<mirage::type::FPType *>(dmm->fp_base_ptr[gpu_id] +
                                                 input_tensors[0].fp_offset);
    mirage::type::FPType *output_fp_ptr =
        reinterpret_cast<mirage::type::FPType *>(dmm->fp_base_ptr[gpu_id] +
                                                 output_tensors[0].fp_offset);
    compute_rms_norm_fingerprint<<<num_blocks, num_threads_per_blk>>>(
        input_fp_ptr,
        output_fp_ptr,
        dmm->div_p_lookup_table,
        dmm->div_q_lookup_table,
        dmm->sqrt_p_lookup_table,
        dmm->sqrt_q_lookup_table,
        num_samples,
        normalized_size);
    checkCUDA(hipDeviceSynchronize());
  }
  return true;
}

} // namespace kernel
} // namespace mirage
