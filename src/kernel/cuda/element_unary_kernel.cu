#include "hip/hip_runtime.h"
/* Copyright 2023-2024 CMU
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cutlass/fast_math.h"
#include "mirage/config.h"
#include "mirage/kernel/device_memory_manager.h"
#include "mirage/kernel/element_unary.h"
#include "mirage/kernel/graph.h"
#include "mirage/utils/cuda_helper.h"
#include "mirage/utils/hash_utils.h"
#include <cassert>

namespace mirage {
namespace kernel {

using namespace mirage::type;
using namespace mirage::config;

template <typename DT>
__global__ void execute_elementunary(mirage::type::KNOperatorType type,
                                     DT *input_ptr,
                                     DT *output_ptr,
                                     int num_elements) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (type == mirage::type::KN_EXP_OP) {
    if (i < num_elements) {
      output_ptr[i] = cutlass::fast_exp(input_ptr[i]);
    }
  } else if (type == mirage::type::KN_SILU_OP) {
    if (i < num_elements) {
      DT x = input_ptr[i];
      output_ptr[i] = x / (1.0f + cutlass::fast_exp(-x));
    }
  } else {
    assert(false && "Unimplemented");
  }
}

bool KNElementUnaryOp::profile(ProfileResult &result) {
  // Only launch kernel on a single GPU for profiling
  checkCUDA(hipSetDevice(0));
  assert(input_tensors[0].num_elements() == output_tensors[0].num_elements());
  assert(input_tensors[0].data_type == DT_FLOAT16);
  assert(output_tensors[0].data_type == DT_FLOAT16);
  mirage::kernel::DeviceMemoryManager *dmm =
      mirage::kernel::DeviceMemoryManager::get_instance();
  cutlass::half_t *input_ptr = reinterpret_cast<cutlass::half_t *>(
      dmm->data_base_ptr[0] + input_tensors[0].data_offset);
  cutlass::half_t *output_ptr = reinterpret_cast<cutlass::half_t *>(
      dmm->data_base_ptr[0] + output_tensors[0].data_offset);
  int num_elements = input_tensors[0].num_elements();
  int const num_threads_per_blk = 1024;
  int num_blocks =
      (num_elements + num_threads_per_blk - 1) / num_threads_per_blk;
  checkCUDA(hipDeviceSynchronize());
  hipEvent_t events[2];
  checkCUDA(hipEventCreate(&events[0]));
  checkCUDA(hipEventCreate(&events[1]));
  checkCUDA(hipEventRecord(events[0]));
  for (int i = 0; i < ProfileResult::NUM_ITERATIONS; i++) {
    execute_elementunary<<<num_blocks, num_threads_per_blk>>>(
        op_type, input_ptr, output_ptr, num_elements);
  }
  float runtime_ms = 0;
  checkCUDA(hipEventRecord(events[1]));
  checkCUDA(hipEventSynchronize(events[1]));
  checkCUDA(hipEventElapsedTime(&runtime_ms, events[0], events[1]));
  result.run_time = runtime_ms / ProfileResult::NUM_ITERATIONS;
  printf("ElementUnary: runtime(%.8lfms)\n", result.run_time);
  checkCUDA(hipEventDestroy(events[0]));
  checkCUDA(hipEventDestroy(events[1]));
  return true;
}

__global__ void
    compute_elementunary_fingerprint(mirage::type::KNOperatorType type,
                                     FPType *exp_lookup_table,
                                     mirage::type::FPType *input_ptr,
                                     mirage::type::FPType *output_ptr,
                                     int num_elements) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (type == mirage::type::KN_EXP_OP) {
    if (i < num_elements) {
      mirage::type::FPType val = input_ptr[i];
      mirage::type::FPType q_residual = val % FP_Q;
      uint32_t result = exp_lookup_table[q_residual];
      result = (result * FP_Q_MUL_P_MOD_1) % FP_PQ;
      output_ptr[i] = result;
    }
  } else if (type == mirage::type::KN_SILU_OP) {
    if (i < num_elements) {
      output_ptr[i] = compute_silu_fingerprint(input_ptr[i], exp_lookup_table);
    }
  } else {
    assert(false && "Unimplemented");
  }
}

bool KNElementUnaryOp::fingerprint(void) {
  // assert a 1-D GPU mesh
  assert(kgraph->gpu_dim.y == 1);
  assert(kgraph->gpu_dim.z == 1);
  assert(input_tensors[0].num_elements() == output_tensors[0].num_elements());
  int num_elements = input_tensors[0].num_elements();
  int const num_threads_per_blk = 1024;
  int num_blocks =
      (num_elements + num_threads_per_blk - 1) / num_threads_per_blk;
  mirage::kernel::DeviceMemoryManager *dmm =
      mirage::kernel::DeviceMemoryManager::get_instance();
  // Use GPU 0 for computing fingerprint
  checkCUDA(hipSetDevice(0));
  for (int gpu_id = 0; gpu_id < kgraph->gpu_dim.x; gpu_id++) {
    mirage::type::FPType *input_fp_ptr =
        reinterpret_cast<mirage::type::FPType *>(dmm->fp_base_ptr[gpu_id] +
                                                 input_tensors[0].fp_offset);
    mirage::type::FPType *output_fp_ptr =
        reinterpret_cast<mirage::type::FPType *>(dmm->fp_base_ptr[gpu_id] +
                                                 output_tensors[0].fp_offset);
    compute_elementunary_fingerprint<<<num_blocks, num_threads_per_blk>>>(
        op_type,
        dmm->exp_lookup_table,
        input_fp_ptr,
        output_fp_ptr,
        num_elements);
    checkCUDA(hipDeviceSynchronize());
  }
  return true;
}

} // namespace kernel
} // namespace mirage
