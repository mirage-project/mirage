#include "hip/hip_runtime.h"
#include "cutlass/fast_math.h"
#include "mirage/config.h"
#include "mirage/kernel/chunk.h"
#include "mirage/kernel/device_memory_manager.h"
#include "mirage/kernel/graph.h"
#include "mirage/utils/cuda_helper.h"
#include "mirage/utils/fingerprint_functions.h"
#include "mirage/utils/hash_utils.h"
#include <cassert>
#include <iostream>

namespace mirage {
namespace kernel {

using namespace mirage::type;
using namespace mirage::config;
using namespace mirage::utils;

template <typename DT>
__global__ void execute_chunk(DT *input_ptr,
                              DT *output1_ptr,
                              DT *output2_ptr,
                              int3 input_shape,
                              int3 output_shape,
                              int chunk_size,
                              int chunk_dim,
                              int num_elements) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < num_elements) {
    int input_i = i / (input_shape.y * input_shape.z);
    int input_j = (i % (input_shape.y * input_shape.z)) / input_shape.z;
    int input_k = i % input_shape.z;
    if (chunk_dim == 0) {
      if (input_i < output_shape.x) {
        output1_ptr[i] = input_ptr[i];
      } else {
        int i2 =
            ((input_i - output_shape.x) * (output_shape.y * output_shape.z)) +
            (input_j * output_shape.z) + input_k;
        output2_ptr[i2] = input_ptr[i];
      }
    } else if (chunk_dim == 1) {
      if (input_j < output_shape.y) {
        output1_ptr[i] = input_ptr[i];
      } else {
        int i2 = (input_i * (output_shape.y * output_shape.z)) +
                 ((input_j - output_shape.y) * output_shape.z) + input_k;
        output2_ptr[i2] = input_ptr[i];
      }
    } else if (chunk_dim == 2) {
      if (input_k < output_shape.z) {
        output1_ptr[i] = input_ptr[i];
      } else {
        int i2 = (input_i * (output_shape.y * output_shape.z)) +
                 (input_j * output_shape.z) + (input_k - output_shape.z);
        output2_ptr[i2] = input_ptr[i];
      }
    } else { // chunk_dim == 3
      assert(false && "unimplemented");
    }
  }
}

bool KNChunkOp::profile(ProfileResult &result) {
  checkCUDA(hipSetDevice(0));
  assert(input_tensors[0].data_type == DT_FLOAT16);
  assert(output_tensors[1].data_type == DT_FLOAT16);
  assert(output_tensors[2].data_type == DT_FLOAT16);
  mirage::kernel::DeviceMemoryManager *dmm =
      mirage::kernel::DeviceMemoryManager::get_instance();
  cutlass::half_t *input_ptr = reinterpret_cast<cutlass::half_t *>(
      dmm->data_base_ptr[0] + input_tensors[0].data_offset);
  cutlass::half_t *output1_ptr = reinterpret_cast<cutlass::half_t *>(
      dmm->data_base_ptr[0] + output_tensors[0].data_offset);
  cutlass::half_t *output2_ptr = reinterpret_cast<cutlass::half_t *>(
      dmm->data_base_ptr[0] + output_tensors[1].data_offset);

  int num_elements = input_tensors[0].num_elements();
  int3 input_shape = {input_tensors[0].dim[0],
                      input_tensors[0].dim[1],
                      input_tensors[0].dim[2]};
  int3 output_shape = {
      chunk_dim == 0 ? input_shape.x / chunk_size : input_shape.x,
      chunk_dim == 1 ? input_shape.y / chunk_size : input_shape.y,
      chunk_dim == 2 ? input_shape.z / chunk_size : input_shape.z};
  int const num_threads_per_blk = 1024;
  int num_blocks =
      (num_elements + num_threads_per_blk - 1) / num_threads_per_blk;
  checkCUDA(hipDeviceSynchronize());
  hipEvent_t events[2];
  checkCUDA(hipEventCreate(&events[0]));
  checkCUDA(hipEventCreate(&events[1]));
  checkCUDA(hipEventRecord(events[0]));
  for (int i = 0; i < 16; i++) {
    execute_chunk<<<num_blocks, num_threads_per_blk>>>(input_ptr,
                                                       output1_ptr,
                                                       output2_ptr,
                                                       input_shape,
                                                       output_shape,
                                                       chunk_size,
                                                       chunk_dim,
                                                       num_elements);
  }
  float runtime_ms = 0;
  checkCUDA(hipEventRecord(events[1]));
  checkCUDA(hipEventSynchronize(events[1]));
  checkCUDA(hipEventElapsedTime(&runtime_ms, events[0], events[1]));
  result.run_time = runtime_ms / 16;
  printf("Chunk: runtime(%.8lfms)\n", result.run_time);
  checkCUDA(hipEventDestroy(events[0]));
  checkCUDA(hipEventDestroy(events[1]));
  return true;
}

__global__ void compute_chunk_fingerprint(char *dmem_fp_ptr,
                                          mirage::kernel::DTensor input,
                                          mirage::kernel::DTensor output1,
                                          mirage::kernel::DTensor output2,
                                          int3 input_shape,
                                          int3 output_shape,
                                          int chunk_size,
                                          int chunk_dim,
                                          int num_elements) {
  mirage::type::FPType *input_fp_ptr =
      reinterpret_cast<mirage::type::FPType *>(dmem_fp_ptr + input.fp_offset);
  mirage::type::FPType *output1_fp_ptr =
      reinterpret_cast<mirage::type::FPType *>(dmem_fp_ptr + output1.fp_offset);
  mirage::type::FPType *output2_fp_ptr =
      reinterpret_cast<mirage::type::FPType *>(dmem_fp_ptr + output2.fp_offset);

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < num_elements) {
    int input_i = i / (input_shape.y * input_shape.z);
    int input_j = (i % (input_shape.y * input_shape.z)) / input_shape.z;
    int input_k = i % input_shape.z;
    if (chunk_dim == 0) {
      if (input_i < output_shape.x) {
        output1_fp_ptr[i] = input_fp_ptr[i];
      } else {
        int i2 =
            ((input_i - output_shape.x) * (output_shape.y * output_shape.z)) +
            (input_j * output_shape.z) + input_k;
        output2_fp_ptr[i2] = input_fp_ptr[i];
      }
    } else if (chunk_dim == 1) {
      if (input_j < output_shape.y) {
        output1_fp_ptr[i] = input_fp_ptr[i];
      } else {
        int i2 = (input_i * (output_shape.y * output_shape.z)) +
                 ((input_j - output_shape.y) * output_shape.z) + input_k;
        output2_fp_ptr[i2] = input_fp_ptr[i];
      }
    } else if (chunk_dim == 2) {
      if (input_k < output_shape.z) {
        // printf("0: i=%d, coords=(%d, %d, %d)\n", i, input_i, input_j, input_k);
        output1_fp_ptr[i] = input_fp_ptr[i];
      } else {
        int i2 = (input_i * (output_shape.y * output_shape.z)) +
                 (input_j * output_shape.z) + (input_k - output_shape.z);
        // printf("1: i=%d, i2=%d, coords=(%d, %d, %d)\n", i, i2, input_i, input_j, input_k);
        output2_fp_ptr[i2] = input_fp_ptr[i];
      }
    } else { // chunk_dim == 3
      assert(false && "unimplemented");
    }
  }
}

bool KNChunkOp::fingerprint(void) {
  assert(kgraph->gpu_dim.y == 1);
  assert(kgraph->gpu_dim.z == 1);

  assert(input_tensors[0].num_dims == output_tensors[0].num_dims);
  assert(input_tensors[0].num_dims == output_tensors[1].num_dims);

  int num_elements = input_tensors[0].num_elements();
  int3 input_shape;
  if (input_tensors[0].num_dims == 1) {
    input_shape.x = 1;
    input_shape.y = 1;
    input_shape.z = input_tensors[0].dim[0];
  } else if (input_tensors[0].num_dims == 2) {
    input_shape.x = 1;
    input_shape.y = input_tensors[0].dim[0];
    input_shape.z = input_tensors[0].dim[1];
  } else { // num_dims = 3
    input_shape.x = input_tensors[0].dim[0];
    input_shape.y = input_tensors[0].dim[1];
    input_shape.z = input_tensors[0].dim[2];
  }

  int adjusted_chunk_dim = chunk_dim + (3 - input_tensors[0].num_dims);
  int3 output_shape;
  output_shape.x = adjusted_chunk_dim == 0 ? input_shape.x / chunk_size : input_shape.x;
  output_shape.y = adjusted_chunk_dim == 1 ? input_shape.y / chunk_size : input_shape.y;
  output_shape.z = adjusted_chunk_dim == 2 ? input_shape.z / chunk_size : input_shape.z;

  int const num_threads_per_blk = 1024;
  int num_blocks =
      (num_elements + num_threads_per_blk - 1) / num_threads_per_blk;

  mirage::kernel::DeviceMemoryManager *dmm =
      mirage::kernel::DeviceMemoryManager::get_instance();
  // Use GPU 0 for computing fingerprint
  checkCUDA(hipSetDevice(0));
  for (int gpu_id = 0; gpu_id < kgraph->gpu_dim.x; gpu_id++) {
    compute_chunk_fingerprint<<<num_blocks, num_threads_per_blk>>>(
        dmm->fp_base_ptr[gpu_id],
        input_tensors[0],
        output_tensors[0],
        output_tensors[1],
        input_shape,
        output_shape,
        chunk_size,
        adjusted_chunk_dim,
        num_elements);
    checkCUDA(hipDeviceSynchronize());
  }
  return true;
}

} // namespace kernel
} // namespace mirage