#include "hip/hip_runtime.h"
/* Copyright 2023-2024 CMU
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cutlass/fast_math.h"
#include "mirage/kernel/device_memory_manager.h"
#include "mirage/kernel/graph.h"
#include "mirage/kernel/reduction.h"
#include "mirage/utils/cuda_helper.h"
#include "mirage/utils/hash_utils.h"
#include <cassert>

namespace mirage {
namespace kernel {

using namespace mirage::type;
using namespace mirage::config;

template <typename DT>
__global__ void execute_reduction(DT *input_ptr,
                                  DT *output_ptr,
                                  int num_input_elements,
                                  int num_output_elements) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  DT sum = static_cast<DT>(0.0f);
  if (idx < num_output_elements) {
    for (int i = 0; i < num_input_elements; i += num_output_elements) {
      sum += input_ptr[i];
    }
    output_ptr[idx] = sum;
  }
}

bool KNReductionOp::profile(ProfileResult &result) {
  // assert a single GPU
  // assert(kgraph->gpu_dim.x == 1);
  int gpu_id = 0;
  checkCUDA(hipSetDevice(0));

  assert(input_tensors[0].data_type == DT_FLOAT16);
  assert(output_tensors[0].data_type == DT_FLOAT16);
  mirage::kernel::DeviceMemoryManager *dmm =
      mirage::kernel::DeviceMemoryManager::get_instance();
  cutlass::half_t *input_ptr = reinterpret_cast<cutlass::half_t *>(
      dmm->data_base_ptr[gpu_id] + input_tensors[0].data_offset);
  cutlass::half_t *output_ptr = reinterpret_cast<cutlass::half_t *>(
      dmm->data_base_ptr[gpu_id] + output_tensors[0].data_offset);
  int num_input_elements = input_tensors[0].num_elements();
  int num_output_elements = output_tensors[0].num_elements();
  int const num_threads_per_blk = 1024;
  int num_blocks =
      (num_output_elements + num_threads_per_blk - 1) / num_threads_per_blk;
  checkCUDA(hipDeviceSynchronize());
  hipEvent_t events[2];
  checkCUDA(hipEventCreate(&events[0]));
  checkCUDA(hipEventCreate(&events[1]));
  checkCUDA(hipEventRecord(events[0]));
  for (int i = 0; i < ProfileResult::NUM_ITERATIONS; i++) {
    execute_reduction<<<num_blocks, num_threads_per_blk>>>(
        input_ptr, output_ptr, num_input_elements, num_output_elements);
  }
  float runtime_ms = 0;
  checkCUDA(hipEventRecord(events[1]));
  checkCUDA(hipEventSynchronize(events[1]));
  checkCUDA(hipEventElapsedTime(&runtime_ms, events[0], events[1]));
  result.run_time = runtime_ms / ProfileResult::NUM_ITERATIONS;
  printf("Reduction: runtime(%.8lfms)\n", result.run_time);
  checkCUDA(hipEventDestroy(events[0]));
  checkCUDA(hipEventDestroy(events[1]));

  return true;
}

__global__ void compute_reduction_fingerprint(FPType *input_ptr,
                                              FPType *output_ptr,
                                              int num_elements,
                                              int reduction_factor,
                                              int input_stride,
                                              int output_stride) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < num_elements) {
    uint32_t result = 0;
    int n = i / output_stride;
    int m = i % output_stride;
    for (int k = 0; k < reduction_factor; k++) {
      result = (result + input_ptr[n * input_stride + m + k * output_stride]) %
               FP_PQ;
      if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
        // printf("result(%d) output_stride(%d) input_stride(%d) i(%d), n(%d) "
        //        "m(%d) k(%d)\n",
        //        result,
        //        output_stride,
        //        input_stride,
        //        i,
        //        n,
        //        m,
        //        k);
      }
    }
    output_ptr[i] = result;
  }
}

bool KNReductionOp::fingerprint(void) {
  // assert a 1-D GPU mesh
  assert(kgraph->gpu_dim.y == 1);
  assert(kgraph->gpu_dim.z == 1);

  int num_elements = output_tensors[0].num_elements();
  int const num_threads_per_blk = 1024;
  int num_blocks =
      (num_elements + num_threads_per_blk - 1) / num_threads_per_blk;
  int output_stride = 1;
  int input_stride = 1;
  for (int i = reduction_dim_idx; i < output_tensors[0].num_dims; i++) {
    output_stride *= output_tensors[0].dim[i];
    input_stride *= input_tensors[0].dim[i];
  }
  int reduction_factor = input_tensors[0].dim[reduction_dim_idx] /
                         output_tensors[0].dim[reduction_dim_idx];
  assert(output_stride * reduction_factor == input_stride);
  mirage::kernel::DeviceMemoryManager *dmm =
      mirage::kernel::DeviceMemoryManager::get_instance();
  // Use GPU 0 for computing fingerprint
  checkCUDA(hipSetDevice(0));
  for (int gpu_id = 0; gpu_id < kgraph->gpu_dim.x; gpu_id++) {
    mirage::type::FPType *input_fp_ptr =
        reinterpret_cast<mirage::type::FPType *>(dmm->fp_base_ptr[gpu_id] +
                                                 input_tensors[0].fp_offset);
    mirage::type::FPType *output_fp_ptr =
        reinterpret_cast<mirage::type::FPType *>(dmm->fp_base_ptr[gpu_id] +
                                                 output_tensors[0].fp_offset);
    compute_reduction_fingerprint<<<num_blocks, num_threads_per_blk>>>(
        input_fp_ptr,
        output_fp_ptr,
        num_elements,
        reduction_factor,
        input_stride,
        output_stride);
    checkCUDA(hipDeviceSynchronize());
  }
  return true;
}

} // namespace kernel
} // namespace mirage
