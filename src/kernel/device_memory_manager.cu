#include "hip/hip_runtime.h"
/* Copyright 2023-2024 CMU
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "mirage/kernel/device_memory_manager.h"
#include "mirage/utils/cuda_helper.h"

namespace mirage {
namespace kernel {

using namespace mirage::type;
using namespace mirage::config;

DeviceMemoryManager *DeviceMemoryManager::singleton = nullptr;

DeviceMemoryManager::DeviceMemoryManager(int _num_gpus) : num_gpus(_num_gpus) {
  // fingerprint related fields
  checkCUDA(hipSetDevice(0));
  // Part 1: exponential lookup table
  // make future tensors 16 bytes aligned
  checkCUDA(
      hipMalloc(&exp_lookup_table, (sizeof(FPType) * FP_Q + 15) / 16 * 16));
  // check PQ relations
  assert(FP_Q < FP_P);
  assert((FP_P - 1) % FP_Q == 0);
  FPType exp_table[FP_Q];
  exp_table[0] = 1;
  for (int i = 1; i < FP_Q; i++) {
    exp_table[i] = (exp_table[i - 1] * FP_EXP_BASE) % FP_P;
  }
  assert((exp_table[FP_Q - 1] * FP_EXP_BASE) % FP_P == 1);
  checkCUDA(hipMemcpy(exp_lookup_table,
                       exp_table,
                       sizeof(FPType) * FP_Q,
                       hipMemcpyHostToDevice));
  // Part 2: division p lookup table
  // make future tensors 16 bytes aligned
  checkCUDA(
      hipMalloc(&div_p_lookup_table, (sizeof(FPType) * FP_P + 15) / 16 * 16));
  FPType div_p_table[FP_P];
  for (uint32_t i = 0; i < FP_P; i++) {
    div_p_table[i] = 1;
    for (uint32_t j = 1; j < FP_P; j++) {
      if ((i * j) % FP_P == 1) {
        div_p_table[i] = j;
      }
    }
    if (i > 1) {
      assert(div_p_table[i] != 1);
    }
  }
  checkCUDA(hipMemcpy(div_p_lookup_table,
                       div_p_table,
                       sizeof(FPType) * FP_P,
                       hipMemcpyHostToDevice));
  // Part 3: division q lookup table
  // make future tensors 16 bytes aligned
  checkCUDA(
      hipMalloc(&div_q_lookup_table, (sizeof(FPType) * FP_Q + 15) / 16 * 16));
  FPType div_q_table[FP_Q];
  for (uint32_t i = 0; i < FP_Q; i++) {
    div_q_table[i] = 1;
    for (uint32_t j = 1; j < FP_Q; j++) {
      if ((i * j) % FP_Q == 1) {
        div_q_table[i] = j;
      }
    }
    if (i > 1) {
      assert(div_q_table[i] != 1);
    }
  }
  checkCUDA(hipMemcpy(div_q_lookup_table,
                       div_q_table,
                       sizeof(FPType) * FP_Q,
                       hipMemcpyHostToDevice));
  // Part 4: sqrt p lookup table
  // make future tensors 16 bytes aligned
  checkCUDA(
      hipMalloc(&sqrt_p_lookup_table, (sizeof(FPType) * FP_P + 15) / 16 * 16));
  // Solving the congruence b=x^2 mod p using the following formulas:
  // if p == 3 mod 4, then x = b^{(p+1)/4} is a solution
  assert(FP_P % 4 == 3);
  FPType sqrt_p_table[FP_P];
  for (uint32_t i = 0; i < FP_P; i++) {
    sqrt_p_table[i] = 1;
    for (uint32_t j = 0; j < (FP_P + 1) / 4; j++) {
      sqrt_p_table[i] = (sqrt_p_table[i] * i) % FP_P;
    }
    // assert((sqrt_p_table[i] * sqrt_p_table[i]) % FP_P == i);
  }
  checkCUDA(hipMemcpy(sqrt_p_lookup_table,
                       sqrt_p_table,
                       sizeof(FPType) * FP_P,
                       hipMemcpyHostToDevice));
  // Part 5: sqrt q lookup table
  // make future tensors 16 bytes aligned
  checkCUDA(
      hipMalloc(&sqrt_q_lookup_table, (sizeof(FPType) * FP_Q + 15) / 16 * 16));
  assert(FP_Q % 4 == 3);
  FPType sqrt_q_table[FP_Q];
  for (uint32_t i = 0; i < FP_Q; i++) {
    sqrt_q_table[i] = 1;
    for (uint32_t j = 0; j < (FP_Q + 1) / 4; j++) {
      sqrt_q_table[i] = (sqrt_q_table[i] * i) % FP_Q;
    }
    // assert((sqrt_q_table[i] * sqrt_q_table[i]) % FP_Q == i);
  }
  checkCUDA(hipMemcpy(sqrt_q_lookup_table,
                       sqrt_q_table,
                       sizeof(FPType) * FP_Q,
                       hipMemcpyHostToDevice));
  // data and fingerprints
  for (int i = 0; i < num_gpus; i++) {
    checkCUDA(hipSetDevice(i));
    checkCUDA(hipStreamCreate(&stream[i]));
    checkCUDA(
        hipMalloc(&data_base_ptr[i], mirage::config::MAX_DMEM_DATA_SIZE));
    checkCUDA(hipblasCreate(&blas[i]));
    checkCUDA(hipblasSetMathMode(blas[i], HIPBLAS_TENSOR_OP_MATH));
    // Note that we allocate all fingerprint buffers
    // on the 0-th GPU to avoid inter-GPU communication
    // for computing fingerprints
    // In addition, we allocate an extra space for storing
    // stensors' fingerprints in the device memory
    if (i == 0) {
      for (int k = 0; k < num_gpus; k++) {
        checkCUDA(
            hipMalloc(&fp_base_ptr[k], mirage::config::MAX_DMEM_FP_SIZE));
      }
      checkCUDA(
          hipMalloc(&stensor_fp_base_ptr,
                     mirage::config::MAX_SMEM_FP_SIZE *
                         mirage::config::MAX_NUM_THREADBLOCKS_PER_KERNEL));
    }
  }
}

DeviceMemoryManager::~DeviceMemoryManager() {
  for (int i = 0; i < num_gpus; i++) {
    hipSetDevice(i);
    checkCUDA(hipFree(data_base_ptr[i]));
    checkCUDA(hipStreamDestroy(stream[i]));
    checkCUDA(hipblasDestroy(blas[i]));
    if (i == 0) {
      checkCUDA(hipFree(exp_lookup_table));
      checkCUDA(hipFree(div_p_lookup_table));
      checkCUDA(hipFree(div_q_lookup_table));
      for (int k = 0; k < num_gpus; k++) {
        checkCUDA(hipFree(fp_base_ptr[i]));
      }
      checkCUDA(hipFree(stensor_fp_base_ptr));
    }
  }
}

#ifdef DEADCODE
bool DeviceMemoryManager::allocate(DTensor &tensor, bool allocate_fingerprint) {
  // assert that the start of the tensor is 16 bytes aligned
  assert(offset % 16 == 0);
  char *ret_ptr = base_ptr + offset;
  size_t tensor_size = tensor.data_size();
  // make tensor_size a multiplier of 16
  tensor_size = (tensor_size + 15) / 16 * 16;
  offset += tensor_size;
  tensor.data_offset = ret_ptr - base_ptr;
  allocated_tensors.push_back(std::make_pair(tensor.data_offset, tensor_size));

  if (allocate_fingerprint) {
    assert(offset % 16 == 0);
    ret_ptr = base_ptr + offset;
    size_t tensor_size = tensor.fingerprint_size();
    tensor_size = (tensor_size + 15) / 16 * 16;
    offset += tensor_size;
    tensor.fp_offset = ret_ptr - base_ptr;
    allocated_tensors.push_back(std::make_pair(tensor.fp_offset, tensor_size));
  }
  // Assert that we haven't used more than what we pre-allocated
  assert(offset <= total_size);

  return true;
}

bool DeviceMemoryManager::free(DTensor &tensor) {
  // Currently assume that tensors are freed in the reverse order
  // so ptr must be the last tensor we have created
  // Note that a non-negative fp_offset means that we have
  // allocated memory for its fingerprint
  if (tensor.fp_offset >= 0) {
    assert(allocated_tensors.size() > 0);
    assert(allocated_tensors.back().first == tensor.fp_offset);
    offset -= allocated_tensors.back().second;
    allocated_tensors.pop_back();
  }
  assert(allocated_tensors.size() > 0);
  assert(allocated_tensors.back().first == tensor.data_offset);
  offset -= allocated_tensors.back().second;
  allocated_tensors.pop_back();
  return true;
}
#endif

DeviceMemoryManager *DeviceMemoryManager::get_instance() {
  if (singleton == nullptr) {
    int num_gpus;
    checkCUDA(hipGetDeviceCount(&num_gpus));
    singleton = new DeviceMemoryManager(num_gpus /*num_gpus*/);
  }
  return singleton;
}

} // namespace kernel
} // namespace mirage
