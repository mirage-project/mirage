#include "hip/hip_runtime.h"
#define NUM_GPUS 1
#define USE_NVSHMEM false
#define MIRAGE_BLACKWELL
#include "runtime.h"
// debug用的头文件
#include <threadblock/utils.h>
#include <random>
#include <cstdlib>
#include <ctime>
#include <cstring>
#include <algorithm>
#include <cmath>
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <cutlass/util/print_error.hpp>
#include <cute/util/debug.hpp>
#include <cutlass/gemm/collective/builders/sm100_common.inl>
// debug用的头文件
using namespace cute;

// 添加参考GEMM实现
template <class AccType,
          class TensorA, class TensorB,
          class TensorC, class TensorD,
          class Alpha, class Beta>
void
reference_gemm(TensorA const& tensor_A, TensorB const& tensor_B,
               TensorC const& tensor_C, TensorD      & tensor_D,
               Alpha alpha, Beta beta)
{
  using namespace cute;
  for (int m = 0; m < size<0>(tensor_D); ++m) {
    for (int n = 0; n < size<1>(tensor_D); ++n) {
      AccType c = AccType(0.f);
      for (int k = 0; k < size<1>(tensor_A); ++k) {
        c += tensor_A(m,k) * tensor_B(n,k);
      }
      tensor_D(m,n) = alpha * c + beta * tensor_C(m,n);
    }
  }
}

// 添加tensor初始化函数
template <class Tensor>
void
initialize_tensor(Tensor& tensor, cute::tuple<int, int> value_range = {-2, 2})
{
  using DataType = typename Tensor::element_type;
  auto [min, max] = value_range;
  for (int i = 0; i < cute::size(tensor); i++) {
    // tensor(i) = DataType(int((max-min)*(rand() / double(RAND_MAX)) + min));
    tensor(i) = DataType(1);
  }
}

template <class TMA_10000003, class TMA_10000004>
__global__ void  __launch_bounds__(384) custom_kernel_0(CUTE_GRID_CONSTANT TMA_10000003 const tma_10000003, CUTE_GRID_CONSTANT TMA_10000004 const tma_10000004,  float* dtensor10000005_ptr, half_t const* dtensor10000003_ptr, half_t const* dtensor10000004_ptr) {
// block x, y is executing here
// if (threadIdx.x == 0) {
//     printf("block x, y is executing here, blockIdx.x is %d, blockIdx.y is %d\n", blockIdx.x, blockIdx.y);
// }

int thread_idx = threadIdx.x;
static constexpr int NUM_THREADS = 128;
static constexpr int CONSUMER_NUM_THREADS = 25;
// UMMA part
// zy: put cluster shape, tiled_mma, mma_tiler here
auto cluster_shape = make_shape(Int<4>{}, Int<4>{}, Int<1>{});
auto tiled_mma = cutlass::gemm::collective::detail::sm100_make_2sm_trivial_tiled_mma<half_t, half_t, float, Shape<Int<256>, Int<256>>, decltype(cluster_shape), UMMA::Major::K, UMMA::Major::K>();
auto mma_tiler = make_shape(tile_size<0>(tiled_mma), tile_size<1>(tiled_mma), tile_size<2>(tiled_mma)*_4{});
// auto tiled_mma = make_tiled_mma(SM100_MMA_F16BF16_2x1SM_SS<half_t, half_t, float, 256, 256, UMMA::Major::K, UMMA::Major::K>{});


// auto cluster_shape = make_shape(Int<4>{}, Int<4>{}, Int<1>{});
uint32_t elect_one_thr  = cute::elect_one_sync();
uint32_t elect_one_warp = (threadIdx.x / 32 == 0); 
Layout cluster_layout_vmnk = tiled_divide(make_layout(cluster_shape), make_tile(typename decltype(tiled_mma)::AtomThrID{}));
// if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
//     printf("cluster_layout_vmnk is: \n");
//     print(cluster_layout_vmnk);
// }
// printf("TiledMMA::AtomThrID{} is: \n");
// print(TiledMMA::AtomThrID{});
int cta_rank = cute::block_rank_in_cluster();
auto cta_in_cluster_coord_vmnk = cluster_layout_vmnk.get_flat_coord(cta_rank);
auto elect_one_cta = get<0>(cta_in_cluster_coord_vmnk) == Int<0>{};
// STensors
extern __shared__ char buf[];
// Remove the separate __shared__ declaration
// __sharedauto__ uint32_t tmem_base_ptr;

// Calculate tmem_base_ptr position at the end of shared memory with 16-byte alignment
// Current smem usage ends around 196800, we need to align to 16 bytes and add space for uint32_t
constexpr size_t TMEM_BASE_PTR_SIZE = sizeof(uint32_t);
constexpr size_t TMEM_BASE_PTR_ALIGNMENT = 16;
constexpr size_t CURRENT_SMEM_USAGE = 196864;  // Current usage
constexpr size_t ALIGNED_TMEM_OFFSET = ((CURRENT_SMEM_USAGE + TMEM_BASE_PTR_ALIGNMENT - 1) / TMEM_BASE_PTR_ALIGNMENT) * TMEM_BASE_PTR_ALIGNMENT;

// Place tmem_base_ptr at the aligned offset
uint32_t* tmem_base_ptr = (uint32_t*)(buf + ALIGNED_TMEM_OFFSET);

using TmemAllocator = cute::TMEM::Allocator2Sm;
TmemAllocator tmem_allocator{};
if (elect_one_warp) { 
  tmem_allocator.allocate(TmemAllocator::Sm100TmemCapacityColumns, tmem_base_ptr);
}
__syncthreads();

float *stensor20000015_ptr = (float*)(buf + 128);
half_t *stensor20000013_ptr = (half_t*)(buf + 65664);
half_t *stensor20000012_ptr = (half_t*)(buf + 128);



// if (threadIdx.x == 0) {
//     printf("block x, y is executing here, blockIdx.x is %d, blockIdx.y is %d\n", blockIdx.x, blockIdx.y);
// }
// G->S copy atoms
// Copy for G->S: dtensor 10000003 -> stensor 20000012
using DTensor10000003TileLayout = Layout<Shape<Int<64>, Int<128>>, Stride<Int<1>, Int<256>>>;
// zy: add atom thr shape based on tiled mma
using AtomThrShapeMNK = Shape<decltype(shape<0>(typename decltype(tiled_mma)::ThrLayoutVMNK{})), _1, _1>;

tb::BlackwellAsyncPipeline<4, decltype(cluster_shape), AtomThrShapeMNK> blackwell_async_pipeline_20000012((void *) (buf + 196736), (tb::warpgroup_id() == 2 && tb::warp_id() % mirage::config::NUM_WARPS_PER_GROUP == 0), tb::warpgroup_id() < 2, 32768, 2, elect_one_cta);
// change zy: use the DstLayout
// using STensor20000012InputAtom = tb::InputTMAAsyncCopy_Blackwell<half_t, decltype(composition(Swizzle<3, 3, 4>{}, Layout<Shape<Int<128>, Int<64>>, Stride<Int<64>, Int<1>>>{})), Layout<Shape<Int<512>, Int<256>>, Stride<Int<256>, Int<1>>>, decltype(tma_10000003), decltype(blackwell_async_pipeline_20000012), true, 4, TiledMMA, MmaTiler_MNK>;
// if (block0() && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
//   printf("DstPipeLayout_10000003: in kernel \n");
//   print(DstPipeLayout_10000003{});
//   printf("\n");
//   printf("\n");
// }

// zy: add is leader check
// using STensor20000012InputAtom = tb::InputTMAAsyncCopy_Blackwell<half_t, decltype(composition(Swizzle<3, 3, 4>{}, Layout<Shape<Int<128>, Int<64>>, Stride<Int<64>, Int<1>>>{})), Layout<Shape<Int<512>, Int<256>>, Stride<Int<256>, Int<1>>>, decltype(tma_10000003), decltype(blackwell_async_pipeline_20000012), true, 4, TiledMMA, MmaTiler_MNK>;
// zy: src layout的stride似乎不重要 只用了shape？
using STensor20000012InputAtom = tb::InputTMAAsyncCopy_Blackwell<half_t, decltype(composition(Swizzle<3, 4, 3>{}, Layout<Shape<Int<128>, Int<64>>, Stride<Int<1>, Int<128>>>{})), Layout<Shape<Int<512>, Int<256>>, Stride<Int<1>, Int<512>>>, decltype(tma_10000003), decltype(blackwell_async_pipeline_20000012), true, 4, decltype(tiled_mma), decltype(mma_tiler), decltype(cluster_shape)>;
// // Copy for G->S: dtensor 10000004 -> stensor 20000013
using DTensor10000004TileLayout = Layout<Shape<Int<256>, Int<64>>, Stride<Int<1>, Int<1024>>>;
tb::BlackwellAsyncPipeline<4, decltype(cluster_shape)> blackwell_async_pipeline_20000013((void *) (buf + 196800), (tb::warpgroup_id() == 2 && tb::warp_id() % mirage::config::NUM_WARPS_PER_GROUP == 0), tb::warpgroup_id() < 2, 32768, 2, elect_one_cta);
using STensor20000013InputAtom = tb::InputTMAAsyncCopy_Blackwell<half_t, decltype(composition(Swizzle<3, 4, 3>{}, Layout<Shape<Int<128>, Int<64>>, Stride<Int<1>, Int<128>>>{})), Layout<Shape<Int<1024>, Int<256>>, Stride<Int<1>, Int<1024>>>, decltype(tma_10000004), decltype(blackwell_async_pipeline_20000013), false, 4, decltype(tiled_mma), decltype(mma_tiler), decltype(cluster_shape)>;



__syncthreads();
  *((uint128_t*)buf) = 0ul;
  
  // S->G copy atoms
  // Copy for S->G: stensor 20000015 -> dtensor 10000005
  // float *dtensor10000005_tile_ptr = dtensor10000005_ptr  + blockIdx.x*128*1024 + blockIdx.y*256*1;
  float *dtensor10000005_tile_ptr = dtensor10000005_ptr;
  using DTensor10000005TileLayout = Layout<Shape<Int<256>, Int<128>>, Stride<Int<1>, Int<1024>>>;
  using STensor20000015OutputAtom = tb::OutputChunkedSyncCopy<float, DTensor10000005TileLayout, Layout<Shape<Int<256>, Int<128>>, Stride<Int<1>, Int<256>>>, NUM_THREADS>;
  
  
  using Matmul20000015LayoutA = decltype(composition(Swizzle<3, 3, 4>{}, Layout<Shape<Int<64>, Int<128>>, Stride<Int<1>, Int<64>>>{}));
  using Matmul20000015LayoutB = decltype(composition(Swizzle<3, 3, 4>{}, Layout<Shape<Int<256>, Int<64>>, Stride<Int<1>, Int<256>>>{}));
  // using Matmul20000015LayoutC = Layout<Shape<Int<256>, Int<128>>, Stride<Int<1>, Int<256>>>;
  using Matmul20000015LayoutC = Layout<Shape<Int<256>, Int<256>>, Stride<Int<1024>, Int<1>>>;
  
  auto mC = make_tensor(make_gmem_ptr<float>(dtensor10000005_ptr), make_layout(make_shape(1024, 1024), make_stride(1024, Int<1>{})));

  // zy: add mc and gC to the kernel
  using Matmul20000015Kernel = tb::Blackwell_Matmul<half_t, true, false, Matmul20000015LayoutA, Matmul20000015LayoutB, Matmul20000015LayoutC, NUM_THREADS, 0, false, true, true, true, 4, decltype(cluster_shape), decltype(tiled_mma), decltype(mma_tiler)>;
  auto matmul_20000015_accum = Matmul20000015Kernel::get_mma_tC(blockIdx.x, blockIdx.y, *tmem_base_ptr);
  

  __syncthreads();

  // if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0) {
  //   printf("\ntmem_base_ptr address: %p, value: 0x%x\n", tmem_base_ptr, *tmem_base_ptr);
  //   printf("matmul_20000015_accum: \n");
  //   print(matmul_20000015_accum);
  // }
  int warpgroup_id = tb::warpgroup_id();

  if (warpgroup_id == 2) {
    if (tb::warp_id_in_wg() == 0) {
      for (uint32_t for_idx = 0; for_idx < 4; for_idx++) {
        STensor20000012InputAtom::run(tma_10000003, stensor20000012_ptr, tiled_mma, mma_tiler, for_idx, blackwell_async_pipeline_20000012);
        STensor20000013InputAtom::run(tma_10000004, stensor20000013_ptr, tiled_mma, mma_tiler, for_idx, blackwell_async_pipeline_20000013);
      }
    }
  }
  else {
    // Consumer main loop
    // zy: add elect_one_cta
      for (uint32_t for_idx = 0; for_idx < 4; for_idx++) {
        // OP type: tb_matmul_op
        {
          if (elect_one_cta) {
            int read_idx_20000012 = blackwell_async_pipeline_20000012.consumer_wait();
            int read_idx_20000013 = blackwell_async_pipeline_20000013.consumer_wait();
            Matmul20000015Kernel::run(matmul_20000015_accum, stensor20000012_ptr, stensor20000013_ptr, for_idx, tiled_mma, read_idx_20000012, blackwell_async_pipeline_20000012, blackwell_async_pipeline_20000013);
          }
        }

    }
    // Write back in-register accumulators
    // tb::wg_sync<CONSUMER_NUM_THREADS>(8);
    // Matmul20000015Kernel::write_tC_to_rC(matmul_20000015_accum, thread_idx);
    // Matmul20000015Kernel::write_back_mma_rC(stensor20000015_ptr, matmul_20000015_accum, thread_idx);
    Matmul20000015Kernel::write_tC_to_gC(stensor20000015_ptr, matmul_20000015_accum, thread_idx, mC);
    // The epilogue (kernels outside the loop)
    // tb::wg_sync<CONSUMER_NUM_THREADS>(8);

    // {
    //   // OP type: tb_output_op
    //   STensor20000015OutputAtom::run(dtensor10000005_tile_ptr, stensor20000015_ptr, thread_idx);
    // }
  }

  // printf("\n finish, threadIdx.x: %d, blockIdx.x: %d, blockIdx.y: %d\n", threadIdx.x, blockIdx.x, blockIdx.y);
  
  cute::cluster_sync();
  if (elect_one_warp) {
    tmem_allocator.release_allocation_lock();
    tmem_allocator.free(*tmem_base_ptr, TmemAllocator::Sm100TmemCapacityColumns);
  }
  // cute::cluster_sync();
}


static void _init() {
}

void _execute_mugraph(std::vector<void const *> input_tensors, std::vector<void*> output_tensors, void* buf, hipStream_t stream, void * profiler_buffer){
  {
    // OP type: kn_input_op
  }
  {
    // OP type: kn_input_op
  }
  {
    // OP type: kn_customized_op
    float *dtensor10000005 = (float*)output_tensors.at(0);  // 更改为float*


    half_t *dtensor10000003 = (half_t*)input_tensors.at(0);
    half_t *dtensor10000004 = (half_t*)input_tensors.at(1);
    dim3 grid_dim(4, 4, 1);
    dim3 block_dim(384, 1, 1);
    size_t smem_size = 196864 + 32;  // Add extra space for tmem_base_ptr with alignment padding
    
    // define tmas
    TiledMMA tiled_mma = cutlass::gemm::collective::detail::sm100_make_2sm_trivial_tiled_mma<half_t, half_t, float, Shape<Int<256>, Int<256>>, decltype(cluster_shape), UMMA::Major::K, UMMA::Major::K>();

    auto cluster_shape = make_shape(Int<4>{}, Int<4>{}, Int<1>{});
    Layout cluster_layout_vmnk = tiled_divide(make_layout(cluster_shape), make_tile(typename decltype(tiled_mma)::AtomThrID{}));
    auto mma_tiler = make_shape(tile_size<0>(tiled_mma), tile_size<1>(tiled_mma), tile_size<2>(tiled_mma)*_4{});
    std::vector<bool> minputs = {true, false};
    
    static constexpr cute::UMMA::Major UMMAMajor_10000003 = UMMA::Major::K;
    using DstMNKLayout_10000003 = decltype(partition_shape_A(tiled_mma, make_shape(size<0>(mma_tiler), size<2>(mma_tiler))));
    using SrcMNKLayout_10000003 = Layout<Shape<Int<512>, Int<256>>, Stride<Int<256>, Int<1>>>;
    using SmemLayoutAtom_10000003 = decltype(cutlass::gemm::collective::detail::sm100_smem_selector<UMMAMajor_10000003, half_t, decltype(get<0>(mma_tiler)), decltype(get<2>(mma_tiler))>());
    // zy: add a stage to last dim
    using DstPipeLayout_10000003 = decltype(UMMA::tile_to_mma_shape(SmemLayoutAtom_10000003{}, append(DstMNKLayout_10000003{}, Int<4>{}), Step<_1,_2,_3>{}));
    auto g_tensor_10000003 = make_tensor(make_gmem_ptr<half_t>(dtensor10000003), SrcMNKLayout_10000003{});
    // printf("g_tensor_10000003: \n");
    // print(g_tensor_10000003);
    // printf("\nDstMNKLayout_10000003: \n");
    // print(DstMNKLayout_10000003{});
    // printf("\nDstPipeLayout_10000003: \n");
    // print(DstPipeLayout_10000003{});
    // printf("\nvalue in g_tensor_10000003: \n");


    // using debugger_10000003 = tb::Debug<DstMNKLayout_10000003, SrcMNKLayout_10000003, SmemLayoutAtom_10000003, DstPipeLayout_10000003, decltype(tiled_mma), decltype(cluster_layout_vmnk), decltype(mma_tiler), decltype(g_tensor_10000003)>;
    // debugger_10000003::run(g_tensor_10000003, tiled_mma, cluster_layout_vmnk, mma_tiler);
    // zy: remember to slice here
    auto tma_10000003 = make_tma_atom_A_sm100(SM100_TMA_2SM_LOAD_MULTICAST{}, g_tensor_10000003, DstPipeLayout_10000003{}(_,_,_,Int<0>{}), mma_tiler, tiled_mma, cluster_layout_vmnk);
    
    static constexpr cute::UMMA::Major UMMAMajor_10000004 = UMMA::Major::K;
    using DstMNKLayout_10000004 = decltype(partition_shape_B(tiled_mma, make_shape(size<1>(mma_tiler), size<2>(mma_tiler))));
    // change to K major for B
    using SrcMNKLayout_10000004 = Layout<Shape<Int<1024>, Int<256>>, Stride<Int<256>, Int<1>>>;
    
    // using SrcMNKLayout_10000004 = Layout<Shape<Int<1024>, Int<256>>, Stride<Int<1>, Int<1024>>>;
    using SmemLayoutAtom_10000004 = decltype(cutlass::gemm::collective::detail::sm100_smem_selector<UMMAMajor_10000004, half_t, decltype(get<1>(mma_tiler)), decltype(get<2>(mma_tiler))>());
    // using DstPipeLayout_10000004 = decltype(UMMA::tile_to_mma_shape(SmemLayoutAtom_10000004{}, (DstMNKLayout_10000004{})));
    using DstPipeLayout_10000004 = decltype(UMMA::tile_to_mma_shape(SmemLayoutAtom_10000004{}, append(DstMNKLayout_10000004{}, Int<4>{}), Step<_1,_2,_3>{}));
    
    auto g_tensor_10000004 = make_tensor(make_gmem_ptr<half_t>(dtensor10000004), SrcMNKLayout_10000004{});
    // using debugger_10000004 = tb::Debug<DstMNKLayout_10000004, SrcMNKLayout_10000004, SmemLayoutAtom_10000004, DstPipeLayout_10000004, decltype(tiled_mma), decltype(cluster_layout_vmnk), decltype(mma_tiler), decltype(g_tensor_10000004)>;
    // debugger_10000004::run(g_tensor_10000004, tiled_mma, cluster_layout_vmnk, mma_tiler);
    // printf("\nDstMNKLayout_10000004: \n");
    // print(DstMNKLayout_10000004{});
    // printf("\nDstPipeLayout_10000004: \n");
    // print(DstPipeLayout_10000004{});
    
    auto tma_10000004 = make_tma_atom_B_sm100(SM100_TMA_2SM_LOAD_MULTICAST{}, g_tensor_10000004, DstPipeLayout_10000004{}(_,_,_,Int<0>{}), mma_tiler, tiled_mma, cluster_layout_vmnk);
    
    // zy: change to add the DstPipeLayout
    auto kernel_ptr = &custom_kernel_0<decltype(tma_10000003), decltype(tma_10000004)>;
    hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_ptr), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size);
    dim3 cluster_dim(size<0>(cluster_shape), size<1>(cluster_shape), size<2>(cluster_shape));
    cutlass::ClusterLaunchParams params = {grid_dim, block_dim, cluster_dim, smem_size};
    cutlass::launch_kernel_on_cluster(params, (void const*) kernel_ptr, tma_10000003, tma_10000004,  dtensor10000005, dtensor10000003, dtensor10000004);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel execution failed: %s\n", hipGetErrorString(err));
        exit(1);
    }
  }
  {
    // OP type: kn_output_op
  }
}

int main() {
    printf("Starting CUDA kernel testing...\n");
    
    // Initialize CUDA
    hipSetDevice(0);
    
    // Set random seed
    srand(time(nullptr));
    
    // A: 512 x 256 (K-major)
    // B: 1024 x 256 (K-major) 
    // D: 512 x 1024 
    const int Gemm_M = 512;
    const int Gemm_N = 1024; 
    const int Gemm_K = 256;
    
    printf("Running GEMM problem size (MxNxK): %dx%dx%d\n", Gemm_M, Gemm_N, Gemm_K);
    
    ////////////////////////////////////////////////////////////
    //
    // Create tensor layouts and data type definitions
    //
    ////////////////////////////////////////////////////////////
    
    // Define data types
    using TypeA = cutlass::half_t; // MMA A data type
    auto type_str_a = "half_t";
    using TypeB = cutlass::half_t; // MMA B data type
    auto type_str_b = "half_t";
    using TypeC = float;           // MMA C data type
    auto type_str_c = "float";
    using TypeD = float;           // MMA D data type
    auto type_str_d = "float";
    using TypeAccumulator = float; // Accumulator type
    
    // A tensor MxK K-major (Layout T = Row-Major)
    Layout layout_A = make_layout(make_shape (Gemm_M,   Gemm_K),
                                  make_stride(Gemm_K, Int<1>{}));   // (Gemm_M,Gemm_K):(Gemm_K,_1)
    // B tensor NxK K-major (Layout N = Column-Major)
    Layout layout_B = make_layout(make_shape (Gemm_N,   Gemm_K),
                                  make_stride(Gemm_K, Int<1>{}));   // (Gemm_N,Gemm_K):(Gemm_K,_1)
    // C tensor MxN N-major (Layout T = Row-Major) 
    Layout layout_C = make_layout(make_shape (Gemm_M,   Gemm_N),
                                  make_stride(Gemm_N, Int<1>{}));   // (Gemm_M,Gemm_N):(Gemm_N,_1)
    // D tensor MxN N-major (Layout T = Row-Major)
    Layout layout_D = make_layout(make_shape (Gemm_M,   Gemm_N),
                                  make_stride(Gemm_N, Int<1>{}));   // (Gemm_M,Gemm_N):(Gemm_N,_1)
    
    ////////////////////////////////////////////////////////////
    //
    // Host memory allocation and tensor creation
    //
    ////////////////////////////////////////////////////////////
    
    // Use thrust for host allocation
    thrust::host_vector<TypeA>   host_A(Gemm_M * Gemm_K);
    Tensor host_tensor_A = make_tensor(host_A.data(), layout_A);
    printf("host_tensor_A:\t"); print(host_tensor_A); printf("\n");
    
    thrust::host_vector<TypeB>   host_B(Gemm_N * Gemm_K);
    Tensor host_tensor_B = make_tensor(host_B.data(), layout_B);
    printf("host_tensor_B:\t"); print(host_tensor_B); printf("\n");
    
    thrust::host_vector<TypeC>   host_C(Gemm_M * Gemm_N);
    Tensor host_tensor_C = make_tensor(host_C.data(), layout_C);
    printf("host_tensor_C:\t"); print(host_tensor_C); printf("\n");
    
    // For storing device output results
    thrust::host_vector<TypeD>   host_D_result(Gemm_M * Gemm_N);
    Tensor host_tensor_D_result = make_tensor(host_D_result.data(), layout_D);
    printf("host_tensor_D_result:\t"); print(host_tensor_D_result); printf("\n");
    
    
    ////////////////////////////////////////////////////////////
    //
    // Initialize tensor data
    //
    ////////////////////////////////////////////////////////////
    

    initialize_tensor(host_tensor_A, make_tuple(-2, 2));
    initialize_tensor(host_tensor_B, make_tuple(-2, 2));
    

    
    // Debug output: check input data
    printf("Sample input data A (first 10 elements): ");
    for(int i = 0; i < 10 && i < size(host_tensor_A); i++) {
        printf("%.4f ", (float)host_tensor_A(i));
    }
    printf("\n");
    printf("Sample input data B (first 10 elements): ");
    for(int i = 0; i < 10 && i < size(host_tensor_B); i++) {
        printf("%.4f ", (float)host_tensor_B(i));
    }
    printf("\n");
    
    ////////////////////////////////////////////////////////////
    //
    // Prepare device memory and execute custom kernel
    //
    ////////////////////////////////////////////////////////////
    
    // Allocate device memory
    thrust::device_vector<TypeA> device_A = host_A;
    thrust::device_vector<TypeB> device_B = host_B;
    thrust::device_vector<TypeC> device_C = host_C;
    thrust::device_vector<TypeD> device_D(Gemm_M * Gemm_N);
    
    Tensor mA = make_tensor(make_gmem_ptr<half_t>(device_A.data().get()), layout_A);
    Tensor mB = make_tensor(make_gmem_ptr<half_t>(device_B.data().get()), layout_B);
    Tensor mC = make_tensor(make_gmem_ptr<float>(device_C.data().get()), layout_C);
    Tensor mD = make_tensor(make_gmem_ptr<float>(device_D.data().get()), layout_D);
    printf("mA:\t"); print(mA); printf("\n");
    printf("mB:\t"); print(mB); printf("\n");
    printf("mC:\t"); print(mC); printf("\n");
    printf("mD:\t"); print(mD); printf("\n");
  
    half_t *d_input1 = device_A.data().get();
    half_t *d_input2 = device_B.data().get();
    float *d_output = device_C.data().get(); 
    printf("d_output:\t"); print(d_output); printf("\n");
    
    
    // Allocate buffers
    const size_t buf_size = 196864 + 1024;
    const size_t profiler_buf_size = 0;
    void *d_buffer, *d_profiler_buffer;
    hipError_t err = hipMalloc(&d_buffer, buf_size);
    if (err != hipSuccess) {
        printf("hipMalloc failed for d_buffer: %s\n", hipGetErrorString(err));
        exit(1);
    }
    err = hipMalloc(&d_profiler_buffer, profiler_buf_size);
    if (err != hipSuccess) {
        printf("hipMalloc failed for d_profiler_buffer: %s\n", hipGetErrorString(err));
        exit(1);
    }
    
    // Prepare input/output vectors
    std::vector<void const *> input_tensors = {d_input1, d_input2};
    std::vector<void*> output_tensors = {d_output, d_output};  // Now float* type
    
    ////////////////////////////////////////////////////////////
    //
    // Performance Testing: Warmup + 2000 runs
    //
    ////////////////////////////////////////////////////////////
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    printf("Starting warmup phase...\n");
    // Warmup: run 50 times
    for(int i = 0; i < 0; i++) {
        
        _execute_mugraph(input_tensors, output_tensors, d_buffer, 0, d_profiler_buffer);
        hipDeviceSynchronize();
    }
    printf("Warmup phase completed.\n");
    
    printf("Starting performance test (2000 runs)...\n");
    // Performance test: run 2000 times and record times
    float total_time = 0.0f;
    std::vector<float> times;
    
    for(int i = 0; i < 1; i++) {

        // Record start time
        hipEventRecord(start);
        
        // Execute kernel
        _execute_mugraph(input_tensors, output_tensors, d_buffer, 0, d_profiler_buffer);
        
        // Record end time
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        
        // Calculate execution time
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        times.push_back(milliseconds);
        total_time += milliseconds;
        
        printf("Run %d: %.4f ms\n", i+1, milliseconds);
    }
    
    // Calculate statistics
    float avg_time = total_time / 2000.0f;
    float min_time = *std::min_element(times.begin(), times.end());
    float max_time = *std::max_element(times.begin(), times.end());
    
    // Calculate standard deviation
    float variance = 0.0f;
    for(float time : times) {
        variance += (time - avg_time) * (time - avg_time);
    }
    variance /= 2000.0f;
    float std_dev = sqrt(variance);
    
    printf("\n=== Mirage Custom Kernel Performance Statistics ===\n");
    printf("GEMM problem size: %dx%dx%d\n", Gemm_M, Gemm_N, Gemm_K);
    printf("Number of runs: 2000\n");
    printf("Average time: %.4f ms\n", avg_time);
    printf("Minimum time: %.4f ms\n", min_time);
    printf("Maximum time: %.4f ms\n", max_time);
    printf("Standard deviation: %.4f ms\n", std_dev);
    printf("Total FLOPS: %.2f GFLOPS\n", (2.0 * Gemm_M * Gemm_N * Gemm_K) / 1e9);
    printf("Performance (based on average time): %.2f GFLOPS\n", (2.0 * Gemm_M * Gemm_N * Gemm_K) / (avg_time * 1e6));
    printf("=======================================\n\n");
    
    // Copy results back to host for verification
    thrust::host_vector<float> temp_output(Gemm_M * Gemm_N);  // Direct use of float
    err = hipMemcpy(temp_output.data(), d_output, Gemm_M * Gemm_N * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("hipMemcpy failed for output: %s\n", hipGetErrorString(err));
        exit(1);
    }
    
    // Direct copy of float results, no conversion needed
    for (int i = 0; i < Gemm_M * Gemm_N; ++i) {
        host_D_result[i] = temp_output[i];
        // if (temp_output[i] != ) {
        //     printf("copy from device to host: %.4f, %.4f\n", temp_output[i], host_D_result[i]);
        // }
    }
    
    printf("Custom kernel execution completed.\n");
    
    ////////////////////////////////////////////////////////////
    //
    // Execute reference GEMM kernel
    //
    ////////////////////////////////////////////////////////////
    
    printf("Executing reference GEMM implementation...\n");
    thrust::host_vector<TypeD> host_reference_D(Gemm_M * Gemm_N);
    auto host_reference_tensor_D = make_tensor(host_reference_D.data(), layout_D);
    
    using Alpha = float;
    using Beta = float;
    Alpha alpha = 1.0f;  // Corresponds to D = alpha * A * B^T + beta * C
    Beta beta = 0.0f;    // Our kernel is actually D = A * B^T
    
    reference_gemm<TypeAccumulator>(host_tensor_A, host_tensor_B, host_tensor_C, host_reference_tensor_D, alpha, beta);
    
    ////////////////////////////////////////////////////////////
    //
    // Compare results
    //
    ////////////////////////////////////////////////////////////
    
    printf("Comparing results...\n");
    auto relative_error = print_matrix_multiply_mollified_relative_error(type_str_a, host_tensor_A,
                                                                         type_str_b, host_tensor_B,
                                                                         type_str_d, host_tensor_D_result, host_reference_tensor_D);
    
    // Print some sample results for debugging
    printf("\nSample results (first 10 elements):\n");
    printf("Actual output: ");
    for(int i = 0; i < Gemm_M*Gemm_N; i++) {
      if (host_D_result[i] != 256) {
        printf("x = %d, y = %d, %.4f ", i%Gemm_M, i/Gemm_M, host_D_result[i]);
        // break;
      }
    }
    printf("\n");
    printf("Reference output: ");
    for(int i = 0; i < 16; i++) {
        printf("%.4f ", host_reference_D[i]);
    }
    printf("\n");
    
    // Success criteria - for half precision, relative error should be small
    bool success = relative_error <= 1e-2;  // 1% tolerance, considering half precision limitations
    printf("Relative error: %.6e\n", relative_error);
    printf("Verification result: %s\n", success ? "PASSED" : "FAILED");
    
    ////////////////////////////////////////////////////////////
    //
    // Clean up resources
    //
    ////////////////////////////////////////////////////////////
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_buffer);
    hipFree(d_profiler_buffer);
    
    printf("\nProgram execution %s.\n", success ? "SUCCESSFUL" : "FAILED");
    return success ? 0 : 1;
} 