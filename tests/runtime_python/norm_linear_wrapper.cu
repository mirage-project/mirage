#include "hip/hip_runtime.h"
#include "norm_linear.cuh"
#include <hip/hip_runtime.h>
#include <torch/extension.h>

using kernel::norm_linear_kernel;

template <typename T>
__global__ void norm_linear_kernel_wrapper(void const *input_ptr,
                                           void const *weight_ptr,
                                           void *output_ptr,
                                           int batch_size,
                                           int hidden_size) {
  norm_linear_kernel<T>(input_ptr, weight_ptr, output_ptr);
}

void launch_norm_linear(torch::Tensor input,
                        torch::Tensor weight,
                        torch::Tensor output) {

  void const *input_ptr = input.data_ptr();
  void const *weight_ptr = weight.data_ptr();
  void *output_ptr = output.data_ptr();

  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  hipFuncSetAttribute(reinterpret_cast<const void*>(norm_linear_kernel_wrapper<__hip_bfloat16>),
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       36666);
  norm_linear_kernel_wrapper<__hip_bfloat16><<<grid_dim, block_dim, 36666>>>(
      input_ptr, weight_ptr, output_ptr, input.size(0), input.size(1));
  hipDeviceSynchronize();
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// pybind11 bindings
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("norm_linear", &launch_norm_linear, "RMSNorm Linear kernel");
}