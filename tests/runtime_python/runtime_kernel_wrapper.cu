#include "hip/hip_runtime.h"
#include "norm_linear.cuh"
#include "silu_mul_linear.cuh"
#include "single_decoding.cuh"
#include <hip/hip_runtime.h>
#include <torch/extension.h>

using kernel::norm_linear_kernel;
using kernel::silu_mul_linear_kernel;
using kernel::single_batch_decoding_kernel;
using bfloat16 = type::bfloat16_t;

template <typename T>
__global__ void single_batch_decoding_kernel_wrapper(void const *qkv_ptr,
                                                     void *k_cache_ptr,
                                                     void *v_cache_ptr,
                                                     void *output_ptr,
                                                     size_t seq_len) {
  single_batch_decoding_kernel<T, 64>(
      qkv_ptr, k_cache_ptr, v_cache_ptr, output_ptr);
}

template <typename T>
__global__ void norm_linear_kernel_wrapper(void const *input_ptr,
                                           void const *weight_ptr,
                                           void *output_ptr) {
  norm_linear_kernel<T>(input_ptr, weight_ptr, output_ptr);
}

template <typename T>
__global__ void silu_mul_linear_kernel_wrapper(void const *input_ptr,
                                               void const *mul_ptr,
                                               void const *weight_ptr,
                                               void *output_ptr) {
  silu_mul_linear_kernel<T>(input_ptr, mul_ptr, weight_ptr, output_ptr);
}

void norm_linear(torch::Tensor input,
                 torch::Tensor weight,
                 torch::Tensor output) {

  void const *input_ptr = input.data_ptr();
  void const *weight_ptr = weight.data_ptr();
  void *output_ptr = output.data_ptr();

  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  hipFuncSetAttribute(reinterpret_cast<const void*>(norm_linear_kernel_wrapper<bfloat16>),
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       36666);
  norm_linear_kernel_wrapper<bfloat16>
      <<<grid_dim, block_dim, 36666>>>(input_ptr, weight_ptr, output_ptr);
  hipDeviceSynchronize();
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

void silu_mul_linear(torch::Tensor input,
                     torch::Tensor mul,
                     torch::Tensor weight,
                     torch::Tensor output) {

  void const *input_ptr = input.data_ptr();
  void const *mul_ptr = mul.data_ptr();
  void const *weight_ptr = weight.data_ptr();
  void *output_ptr = output.data_ptr();

  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  hipFuncSetAttribute(reinterpret_cast<const void*>(silu_mul_linear_kernel_wrapper<bfloat16>),
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       36666);
  silu_mul_linear_kernel_wrapper<bfloat16><<<grid_dim, block_dim, 36666>>>(
      input_ptr, mul_ptr, weight_ptr, output_ptr);
  hipDeviceSynchronize();
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

void single_batch_decoding(torch::Tensor qkv,
                           torch::Tensor k_cache,
                           torch::Tensor v_cache,
                           torch::Tensor output,
                           size_t seq_len) {
  void const *qkv_ptr = qkv.data_ptr();
  void *k_cache_ptr = k_cache.data_ptr();
  void *v_cache_ptr = k_cache.data_ptr();
  void *output_ptr = output.data_ptr();
  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  hipFuncSetAttribute(reinterpret_cast<const void*>(single_batch_decoding_kernel_wrapper<bfloat16>),
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       36666);
  single_batch_decoding_kernel_wrapper<bfloat16>
      <<<grid_dim, block_dim, 36666>>>(
          qkv_ptr, k_cache_ptr, v_cache_ptr, output_ptr, seq_len);
  hipDeviceSynchronize();
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// pybind11 bindings
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("norm_linear", &norm_linear, "RMSNorm Linear kernel");
  m.def("silu_mul_linear", &silu_mul_linear, "SILU MUL Linear kernel");
  m.def("single_batch_decoding",
        &single_batch_decoding,
        "FlashAttention Decoding kernel");
}