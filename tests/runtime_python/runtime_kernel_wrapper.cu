#include "hip/hip_runtime.h"
#include "argmax.cuh"
#include "bfloat16.h"
#include "linear.cuh"
#include "norm_linear.cuh"
#include "paged_attention.cuh"
#include "silu_mul_linear.cuh"
#include "single_batch_decoding.cuh"
#include "single_batch_gqa.cuh"
#include <hip/hip_runtime.h>
#include <torch/extension.h>

// using kernel::argmax_kernel;
using kernel::linear_kernel;
using kernel::norm_linear_task_impl;
using kernel::paged_attention_task_impl;
using kernel::silu_mul_linear_task_impl;
using kernel::single_batch_decoding_kernel;
using kernel::single_batch_gqa_kernel;
using bfloat16 = type::bfloat16_t;

template <typename T>
__global__ void single_batch_gqa_kernel_wrapper(void const *qkv_ptr,
                                                void *k_cache_ptr,
                                                void *v_cache_ptr,
                                                void *output_ptr,
                                                size_t seq_len,
                                                bool qk_norm,
                                                bool rotary_embed,
                                                void const *qnorm_weight_ptr,
                                                void const *knorm_weight_ptr,
                                                void const *cos_ptr,
                                                void const *sin_ptr,
                                                float q_eps,
                                                float k_eps) {
  single_batch_gqa_kernel<T, 4>(qkv_ptr,
                                k_cache_ptr,
                                v_cache_ptr,
                                output_ptr,
                                seq_len,
                                qk_norm,
                                rotary_embed,
                                qnorm_weight_ptr,
                                knorm_weight_ptr,
                                cos_ptr,
                                sin_ptr,
                                q_eps,
                                k_eps);
}

void single_batch_gqa(
    torch::Tensor qkv,
    torch::Tensor k_cache,
    torch::Tensor v_cache,
    torch::Tensor output,
    size_t seq_len,
    bool qk_norm,
    bool rotary_embed,
    torch::optional<torch::Tensor> qnorm_weight = torch::nullopt,
    torch::optional<torch::Tensor> knorm_weight = torch::nullopt,
    torch::optional<torch::Tensor> cos = torch::nullopt,
    torch::optional<torch::Tensor> sin = torch::nullopt,
    float q_eps = 0.0f,
    float k_eps = 0.0f) {
  void const *qkv_ptr = qkv.data_ptr();
  void *k_cache_ptr = k_cache.data_ptr();
  void *v_cache_ptr = v_cache.data_ptr();
  void *output_ptr = output.data_ptr();

  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  size_t smem_size = 88888;

  void const *qnorm_weight_ptr = qk_norm ? qnorm_weight->data_ptr() : nullptr;
  void const *knorm_weight_ptr = qk_norm ? knorm_weight->data_ptr() : nullptr;
  void const *cos_ptr = rotary_embed ? cos->data_ptr() : nullptr;
  void const *sin_ptr = rotary_embed ? sin->data_ptr() : nullptr;

  hipFuncSetAttribute(reinterpret_cast<const void*>(single_batch_gqa_kernel_wrapper<bfloat16>),
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       smem_size);

  single_batch_gqa_kernel_wrapper<bfloat16>
      <<<grid_dim, block_dim, smem_size>>>(qkv_ptr,
                                           k_cache_ptr,
                                           v_cache_ptr,
                                           output_ptr,
                                           seq_len,
                                           qk_norm,
                                           rotary_embed,
                                           qnorm_weight_ptr,
                                           knorm_weight_ptr,
                                           cos_ptr,
                                           sin_ptr,
                                           q_eps,
                                           k_eps);

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// Single Batch Decoding

template <typename T,
          int NUM_Q_HEADS,
          int NUM_KV_HEADS,
          int HEAD_DIM,
          int WEIGHT_STRIDE>
__global__ void single_batch_decoding_wrapper(void const *qkv_ptr,
                                              void *k_cache_ptr,
                                              void *v_cache_ptr,
                                              void *output_ptr,
                                              size_t seq_len,
                                              bool qk_norm,
                                              bool rotary_emd,
                                              void const *qnorm_weight_ptr,
                                              void const *knorm_weight_ptr,
                                              void const *cos_ptr,
                                              void const *sin_ptr,
                                              float q_eps,
                                              float k_eps) {
  single_batch_decoding_kernel<T,
                               NUM_Q_HEADS,
                               NUM_KV_HEADS,
                               HEAD_DIM,
                               WEIGHT_STRIDE>(qkv_ptr,
                                              k_cache_ptr,
                                              v_cache_ptr,
                                              output_ptr,
                                              seq_len,
                                              qk_norm,
                                              rotary_emd,
                                              qnorm_weight_ptr,
                                              knorm_weight_ptr,
                                              cos_ptr,
                                              sin_ptr,
                                              q_eps,
                                              k_eps);
}

void single_batch_decoding(
    torch::Tensor qkv,
    torch::Tensor k_cache,
    torch::Tensor v_cache,
    torch::Tensor output,
    size_t seq_len,
    bool qk_norm,
    bool rotary_emd,
    torch::optional<torch::Tensor> qnorm_weight = torch::nullopt,
    torch::optional<torch::Tensor> knorm_weight = torch::nullopt,
    torch::optional<torch::Tensor> cos = torch::nullopt,
    torch::optional<torch::Tensor> sin = torch::nullopt,
    float q_eps = 0.0f,
    float k_eps = 0.0f) {
  void const *qkv_ptr = qkv.data_ptr();
  void *k_cache_ptr = k_cache.data_ptr();
  void *v_cache_ptr = v_cache.data_ptr();
  void *output_ptr = output.data_ptr();

  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  size_t smem_size = 88888;

  void const *qnorm_weight_ptr = qk_norm ? qnorm_weight->data_ptr() : nullptr;
  void const *knorm_weight_ptr = qk_norm ? knorm_weight->data_ptr() : nullptr;
  void const *cos_ptr = rotary_emd ? cos->data_ptr() : nullptr;
  void const *sin_ptr = rotary_emd ? sin->data_ptr() : nullptr;

  hipFuncSetAttribute(reinterpret_cast<const void*>(single_batch_decoding_wrapper<bfloat16), 4, 1, 128, 128>,
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       smem_size);

  single_batch_decoding_wrapper<bfloat16, 4, 1, 128, 128>
      <<<grid_dim, block_dim, smem_size>>>(qkv_ptr,
                                           k_cache_ptr,
                                           v_cache_ptr,
                                           output_ptr,
                                           seq_len,
                                           qk_norm,
                                           rotary_emd,
                                           qnorm_weight_ptr,
                                           knorm_weight_ptr,
                                           cos_ptr,
                                           sin_ptr,
                                           q_eps,
                                           k_eps);

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// Paged Attention

template <typename T,
          int NUM_Q_PER_KV,
          int HEAD_DIM,
          int PAGE_SIZE,
          int MAX_SEQ_LEN,
          int KV_STRIDE>
__global__ void paged_attention_wrapper(void const *qkv_ptr,
                                        void *paged_k_cache_ptr,
                                        void *paged_v_cache_ptr,
                                        void *output_ptr,
                                        void const *paged_kv_indices_buffer_ptr,
                                        size_t seq_len,
                                        bool qk_norm,
                                        bool rope,
                                        void const *q_norm_weight_ptr,
                                        void const *k_norm_weight_ptr,
                                        void const *cos_ptr,
                                        void const *sin_ptr,
                                        float q_eps,
                                        float k_eps) {
  paged_attention_task_impl<T,
                            NUM_Q_PER_KV,
                            HEAD_DIM,
                            PAGE_SIZE,
                            MAX_SEQ_LEN,
                            KV_STRIDE>(qkv_ptr,
                                       paged_k_cache_ptr,
                                       paged_v_cache_ptr,
                                       output_ptr,
                                       paged_kv_indices_buffer_ptr,
                                       seq_len,
                                       qk_norm,
                                       rope,
                                       q_norm_weight_ptr,
                                       k_norm_weight_ptr,
                                       cos_ptr,
                                       sin_ptr,
                                       q_eps,
                                       k_eps);
}

template <typename T,
          int NUM_Q_PER_KV,
          int HEAD_DIM,
          int PAGE_SIZE,
          int MAX_SEQ_LEN,
          int KV_STRIDE>
void launch_paged_attention(void const *qkv_ptr,
                            void *paged_k_cache_ptr,
                            void *paged_v_cache_ptr,
                            void *output_ptr,
                            void const *paged_kv_indices_buffer_ptr,
                            size_t seq_len,
                            bool qk_norm,
                            bool rope,
                            void const *q_norm_weight_ptr,
                            void const *k_norm_weight_ptr,
                            void const *cos_ptr,
                            void const *sin_ptr,
                            float q_eps,
                            float k_eps) {
  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  size_t smem_size = 112640;

  hipFuncSetAttribute(reinterpret_cast<const void*>(paged_attention_wrapper<T),
                                               NUM_Q_PER_KV,
                                               HEAD_DIM,
                                               PAGE_SIZE,
                                               MAX_SEQ_LEN,
                                               KV_STRIDE>,
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       smem_size);

  paged_attention_wrapper<T,
                          NUM_Q_PER_KV,
                          HEAD_DIM,
                          PAGE_SIZE,
                          MAX_SEQ_LEN,
                          KV_STRIDE>
      <<<grid_dim, block_dim, smem_size>>>(qkv_ptr,
                                           paged_k_cache_ptr,
                                           paged_v_cache_ptr,
                                           output_ptr,
                                           paged_kv_indices_buffer_ptr,
                                           seq_len,
                                           qk_norm,
                                           rope,
                                           q_norm_weight_ptr,
                                           k_norm_weight_ptr,
                                           cos_ptr,
                                           sin_ptr,
                                           q_eps,
                                           k_eps);
}

void paged_attention(
    torch::Tensor qkv,
    torch::Tensor paged_k_cache,
    torch::Tensor paged_v_cache,
    torch::Tensor output,
    torch::Tensor paged_kv_indices_buffer,
    size_t seq_len,
    bool qk_norm,
    bool rope,
    torch::optional<torch::Tensor> q_norm_weight = torch::nullopt,
    torch::optional<torch::Tensor> k_norm_weight = torch::nullopt,
    torch::optional<torch::Tensor> cos = torch::nullopt,
    torch::optional<torch::Tensor> sin = torch::nullopt,
    float q_eps = 0.0f,
    float k_eps = 0.0f) {
  void const *qkv_ptr = qkv.data_ptr();
  void *paged_k_cache_ptr = paged_k_cache.data_ptr();
  void *paged_v_cache_ptr = paged_v_cache.data_ptr();
  void *output_ptr = output.data_ptr();
  void const *paged_kv_indices_buffer_ptr = paged_kv_indices_buffer.data_ptr();

  void const *q_norm_weight_ptr = qk_norm ? q_norm_weight->data_ptr() : nullptr;
  void const *k_norm_weight_ptr = qk_norm ? k_norm_weight->data_ptr() : nullptr;
  void const *cos_ptr = rope ? cos->data_ptr() : nullptr;
  void const *sin_ptr = rope ? sin->data_ptr() : nullptr;

  launch_paged_attention<bfloat16, 4, 128, 64, 512, 128>(
      qkv_ptr,
      paged_k_cache_ptr,
      paged_v_cache_ptr,
      output_ptr,
      paged_kv_indices_buffer_ptr,
      seq_len,
      qk_norm,
      rope,
      q_norm_weight_ptr,
      k_norm_weight_ptr,
      cos_ptr,
      sin_ptr,
      q_eps,
      k_eps);

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// RMSNorm Linear

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
__global__ void norm_linear_kernel_wrapper(void const *input_ptr,
                                           void const *norm_weight_ptr,
                                           void const *weight_ptr,
                                           float eps,
                                           void *output_ptr) {
  norm_linear_task_impl<T,
                        BATCH_SIZE,
                        OUTPUT_SIZE,
                        REDUCTION_SIZE,
                        OUTPUT_SIZE>(
      input_ptr, norm_weight_ptr, weight_ptr, eps, output_ptr);
}

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
void launch_norm_linear(void const *input_ptr,
                        void const *norm_weight_ptr,
                        void const *weight_ptr,
                        float eps,
                        void *output_ptr) {
  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  size_t smem_size = 112640;

  hipFuncSetAttribute(reinterpret_cast<const void*>(
      norm_linear_kernel_wrapper<T), BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>,
      hipFuncAttributeMaxDynamicSharedMemorySize,
      smem_size);

  norm_linear_kernel_wrapper<T, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>
      <<<grid_dim, block_dim, smem_size>>>(
          input_ptr, norm_weight_ptr, weight_ptr, eps, output_ptr);
}

void norm_linear(torch::Tensor input,
                 torch::Tensor norm_weight,
                 torch::Tensor weight,
                 float eps,
                 torch::Tensor output) {

  void const *input_ptr = input.data_ptr();
  void const *norm_weight_ptr = norm_weight.data_ptr();
  void const *weight_ptr = weight.data_ptr();
  void *output_ptr = output.data_ptr();

  switch (output.size(1)) {
    case 16:
      launch_norm_linear<bfloat16, 1, 16, 4096>(
          input_ptr, norm_weight_ptr, weight_ptr, eps, output_ptr);
      break;
    case 32:
      launch_norm_linear<bfloat16, 1, 32, 4096>(
          input_ptr, norm_weight_ptr, weight_ptr, eps, output_ptr);
      break;
    case 64:
      launch_norm_linear<bfloat16, 1, 64, 4096>(
          input_ptr, norm_weight_ptr, weight_ptr, eps, output_ptr);
      break;
    case 256:
      launch_norm_linear<bfloat16, 1, 256, 4096>(
          input_ptr, norm_weight_ptr, weight_ptr, eps, output_ptr);
      break;
    case 1600:
      launch_norm_linear<bfloat16, 1, 1600, 4096>(
          input_ptr, norm_weight_ptr, weight_ptr, eps, output_ptr);
      break;
    default:
      printf("Unsupported output size in test: %zu\n", output.size(1));
      break;
  }

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// SiLU MUL Linear

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
__global__ void silu_mul_linear_kernel_wrapper(void const *input_ptr,
                                               void const *weight_ptr,
                                               void const *bias_ptr,
                                               void *output_ptr) {
  silu_mul_linear_task_impl<T, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>(
      input_ptr, weight_ptr, bias_ptr, output_ptr);
}

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
void launch_silu_mul_linear(void const *input_ptr,
                            void const *weight_ptr,
                            void const *bias_ptr,
                            void *output_ptr) {
  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  size_t smem_size = 112640;

  hipFuncSetAttribute(reinterpret_cast<const void*>(silu_mul_linear_kernel_wrapper<T),
                                                      BATCH_SIZE,
                                                      OUTPUT_SIZE,
                                                      REDUCTION_SIZE>,
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       smem_size);

  silu_mul_linear_kernel_wrapper<T, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>
      <<<grid_dim, block_dim, smem_size>>>(
          input_ptr, weight_ptr, bias_ptr, output_ptr);
}

void silu_mul_linear(torch::Tensor input,
                     torch::Tensor weight,
                     torch::Tensor bias,
                     torch::Tensor output) {

  void const *input_ptr = input.data_ptr();
  void const *weight_ptr = weight.data_ptr();
  void const *bias_ptr = bias.data_ptr();
  void *output_ptr = output.data_ptr();

  switch (output.size(1)) {
    case 16:
      launch_silu_mul_linear<bfloat16, 1, 16, 12288>(
          input_ptr, weight_ptr, bias_ptr, output_ptr);
      break;
    case 32:
      launch_silu_mul_linear<bfloat16, 1, 32, 12288>(
          input_ptr, weight_ptr, bias_ptr, output_ptr);
      break;
    case 64:
      launch_silu_mul_linear<bfloat16, 1, 64, 12288>(
          input_ptr, weight_ptr, bias_ptr, output_ptr);
      break;
    default:
      printf("Unsupported output size in test: %zu\n", output.size(1));
      break;
  }

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// Linear

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
__global__ void linear_kernel_wrapper(void const *input_ptr,
                                      void const *weight_ptr,
                                      void const *residual_ptr,
                                      void *output_ptr) {
  linear_kernel<T, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>(
      input_ptr, weight_ptr, residual_ptr, output_ptr);
}

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
void launch_linear(void const *input_ptr,
                   void const *weight_ptr,
                   void const *residual_ptr,
                   void *output_ptr) {
  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  size_t smem_size = 112640;

  hipFuncSetAttribute(reinterpret_cast<const void*>(
      linear_kernel_wrapper<T), BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>,
      hipFuncAttributeMaxDynamicSharedMemorySize,
      smem_size);

  linear_kernel_wrapper<T, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>
      <<<grid_dim, block_dim, smem_size>>>(
          input_ptr, weight_ptr, residual_ptr, output_ptr);
}

void linear(torch::Tensor input,
            torch::Tensor weight,
            torch::Tensor residual,
            torch::Tensor output) {

  void const *input_ptr = input.data_ptr();
  void const *weight_ptr = weight.data_ptr();
  void const *residual_ptr = residual.data_ptr();
  void *output_ptr = output.data_ptr();

  switch (output.size(1)) {
    case 16:
      launch_linear<bfloat16, 1, 16, 4096>(
          input_ptr, weight_ptr, residual_ptr, output_ptr);
      break;
    case 32:
      launch_linear<bfloat16, 1, 32, 4096>(
          input_ptr, weight_ptr, residual_ptr, output_ptr);
      break;
    case 64:
      launch_linear<bfloat16, 1, 64, 4096>(
          input_ptr, weight_ptr, residual_ptr, output_ptr);
      break;
    default:
      printf("Unsupported output size in test: %zu\n", output.size(1));
      break;
  }

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// Argmax

// template <typename T>
// __global__ void argmax_kernel_wrapper(void const *input_ptr, void
// *output_ptr) {
//   argmax_kernel<T, 1, 32768>(input_ptr, output_ptr);
// }

// template <typename T>
// void launch_argmax(void const *input_ptr, void *output_ptr) {
//   dim3 grid_dim(1, 1, 1);
//   dim3 block_dim(128, 1, 1);
//   size_t smem_size = 36666;

//   hipFuncSetAttribute(reinterpret_cast<const void*>(argmax_kernel_wrapper<T>),
//                        hipFuncAttributeMaxDynamicSharedMemorySize,
//                        smem_size);

//   argmax_kernel_wrapper<T>
//       <<<grid_dim, block_dim, smem_size>>>(input_ptr, output_ptr);
// }

// void argmax(torch::Tensor input, torch::Tensor output) {

//   void const *input_ptr = input.data_ptr();
//   void *output_ptr = output.data_ptr();

//   launch_argmax<bfloat16>(input_ptr, output_ptr);

//   hipError_t err = hipDeviceSynchronize();
//   if (err != hipSuccess) {
//     printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
//   }
// }

// pybind11 bindings

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("linear", &linear, "Linear kernel");
  // m.def("argmax", &argmax, "argmax kernel");
  m.def("norm_linear", &norm_linear, "RMSNorm Linear kernel");
  m.def("silu_mul_linear", &silu_mul_linear, "SILU MUL Linear kernel");
  m.def("single_batch_decoding",
        &single_batch_decoding,
        py::arg("qkv"),
        py::arg("k_cache"),
        py::arg("v_cache"),
        py::arg("output"),
        py::arg("seq_len"),
        py::arg("qk_norm"),
        py::arg("rotary_embed"),
        py::arg("qnorm_weight") = py::none(),
        py::arg("knorm_weight") = py::none(),
        py::arg("cos") = py::none(),
        py::arg("sin") = py::none(),
        py::arg("q_eps") = 0.0f,
        py::arg("k_eps") = 0.0f);
  m.def("paged_attention", &paged_attention, "Paged Attention");
}