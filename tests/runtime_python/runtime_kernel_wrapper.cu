#include "hip/hip_runtime.h"
#include "argmax.cuh"
#include "bfloat16.h"
#include "linear.cuh"
#include "norm.cuh"
#include "norm_linear.cuh"
#include "paged_attention.cuh"
#include "silu_mul_linear.cuh"
#include "single_batch_decoding.cuh"
#include "single_batch_gqa.cuh"
#include <hip/hip_runtime.h>
#include <torch/extension.h>

// using kernel::argmax_kernel;
using kernel::linear_kernel;
using kernel::norm_linear_task_impl;
using kernel::paged_attention_task_impl;
using kernel::silu_mul_linear_task_impl;
using kernel::single_batch_decoding_kernel;
using kernel::single_batch_gqa_kernel;
using bfloat16 = type::bfloat16_t;

template <typename T>
__global__ void single_batch_gqa_kernel_wrapper(void const *qkv_ptr,
                                                void *k_cache_ptr,
                                                void *v_cache_ptr,
                                                void *output_ptr,
                                                size_t seq_len,
                                                bool qk_norm,
                                                bool rotary_embed,
                                                void const *qnorm_weight_ptr,
                                                void const *knorm_weight_ptr,
                                                void const *cos_ptr,
                                                void const *sin_ptr,
                                                float q_eps,
                                                float k_eps) {
  single_batch_gqa_kernel<T, 4>(qkv_ptr,
                                k_cache_ptr,
                                v_cache_ptr,
                                output_ptr,
                                seq_len,
                                qk_norm,
                                rotary_embed,
                                qnorm_weight_ptr,
                                knorm_weight_ptr,
                                cos_ptr,
                                sin_ptr,
                                q_eps,
                                k_eps);
}

void single_batch_gqa(
    torch::Tensor qkv,
    torch::Tensor k_cache,
    torch::Tensor v_cache,
    torch::Tensor output,
    size_t seq_len,
    bool qk_norm,
    bool rotary_embed,
    torch::optional<torch::Tensor> qnorm_weight = torch::nullopt,
    torch::optional<torch::Tensor> knorm_weight = torch::nullopt,
    torch::optional<torch::Tensor> cos = torch::nullopt,
    torch::optional<torch::Tensor> sin = torch::nullopt,
    float q_eps = 0.0f,
    float k_eps = 0.0f) {
  void const *qkv_ptr = qkv.data_ptr();
  void *k_cache_ptr = k_cache.data_ptr();
  void *v_cache_ptr = v_cache.data_ptr();
  void *output_ptr = output.data_ptr();

  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  size_t smem_size = 88888;

  void const *qnorm_weight_ptr = qk_norm ? qnorm_weight->data_ptr() : nullptr;
  void const *knorm_weight_ptr = qk_norm ? knorm_weight->data_ptr() : nullptr;
  void const *cos_ptr = rotary_embed ? cos->data_ptr() : nullptr;
  void const *sin_ptr = rotary_embed ? sin->data_ptr() : nullptr;

  hipFuncSetAttribute(reinterpret_cast<const void*>(single_batch_gqa_kernel_wrapper<bfloat16>),
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       smem_size);

  single_batch_gqa_kernel_wrapper<bfloat16>
      <<<grid_dim, block_dim, smem_size>>>(qkv_ptr,
                                           k_cache_ptr,
                                           v_cache_ptr,
                                           output_ptr,
                                           seq_len,
                                           qk_norm,
                                           rotary_embed,
                                           qnorm_weight_ptr,
                                           knorm_weight_ptr,
                                           cos_ptr,
                                           sin_ptr,
                                           q_eps,
                                           k_eps);

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// Single Batch Decoding

template <typename T,
          int NUM_Q_HEADS,
          int NUM_KV_HEADS,
          int HEAD_DIM,
          int WEIGHT_STRIDE>
__global__ void single_batch_decoding_wrapper(void const *qkv_ptr,
                                              void *k_cache_ptr,
                                              void *v_cache_ptr,
                                              void *output_ptr,
                                              size_t seq_len,
                                              bool qk_norm,
                                              bool rotary_emd,
                                              void const *qnorm_weight_ptr,
                                              void const *knorm_weight_ptr,
                                              void const *cos_ptr,
                                              void const *sin_ptr,
                                              float q_eps,
                                              float k_eps) {
  single_batch_decoding_kernel<T,
                               NUM_Q_HEADS,
                               NUM_KV_HEADS,
                               HEAD_DIM,
                               WEIGHT_STRIDE>(qkv_ptr,
                                              k_cache_ptr,
                                              v_cache_ptr,
                                              output_ptr,
                                              seq_len,
                                              qk_norm,
                                              rotary_emd,
                                              qnorm_weight_ptr,
                                              knorm_weight_ptr,
                                              cos_ptr,
                                              sin_ptr,
                                              q_eps,
                                              k_eps);
}

void single_batch_decoding(
    torch::Tensor qkv,
    torch::Tensor k_cache,
    torch::Tensor v_cache,
    torch::Tensor output,
    size_t seq_len,
    bool qk_norm,
    bool rotary_emd,
    torch::optional<torch::Tensor> qnorm_weight = torch::nullopt,
    torch::optional<torch::Tensor> knorm_weight = torch::nullopt,
    torch::optional<torch::Tensor> cos = torch::nullopt,
    torch::optional<torch::Tensor> sin = torch::nullopt,
    float q_eps = 0.0f,
    float k_eps = 0.0f) {
  void const *qkv_ptr = qkv.data_ptr();
  void *k_cache_ptr = k_cache.data_ptr();
  void *v_cache_ptr = v_cache.data_ptr();
  void *output_ptr = output.data_ptr();

  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  size_t smem_size = 88888;

  void const *qnorm_weight_ptr = qk_norm ? qnorm_weight->data_ptr() : nullptr;
  void const *knorm_weight_ptr = qk_norm ? knorm_weight->data_ptr() : nullptr;
  void const *cos_ptr = rotary_emd ? cos->data_ptr() : nullptr;
  void const *sin_ptr = rotary_emd ? sin->data_ptr() : nullptr;

  hipFuncSetAttribute(reinterpret_cast<const void*>(single_batch_decoding_wrapper<bfloat16), 4, 1, 128, 128>,
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       smem_size);

  single_batch_decoding_wrapper<bfloat16, 4, 1, 128, 128>
      <<<grid_dim, block_dim, smem_size>>>(qkv_ptr,
                                           k_cache_ptr,
                                           v_cache_ptr,
                                           output_ptr,
                                           seq_len,
                                           qk_norm,
                                           rotary_emd,
                                           qnorm_weight_ptr,
                                           knorm_weight_ptr,
                                           cos_ptr,
                                           sin_ptr,
                                           q_eps,
                                           k_eps);

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// Paged Attention

template <typename T,
          int NUM_Q_PER_KV,
          int HEAD_DIM,
          int PAGE_SIZE,
          int MAX_SEQ_LEN,
          int KV_STRIDE>
__global__ void paged_attention_wrapper(void const *qkv_ptr,
                                        void *paged_k_cache_ptr,
                                        void *paged_v_cache_ptr,
                                        void *output_ptr,
                                        void const *paged_kv_indices_buffer_ptr,
                                        size_t seq_len,
                                        bool qk_norm,
                                        bool rope,
                                        void const *q_norm_weight_ptr,
                                        void const *k_norm_weight_ptr,
                                        void const *cos_ptr,
                                        void const *sin_ptr,
                                        float q_eps,
                                        float k_eps) {
  paged_attention_task_impl<T,
                            NUM_Q_PER_KV,
                            HEAD_DIM,
                            PAGE_SIZE,
                            MAX_SEQ_LEN,
                            KV_STRIDE>(qkv_ptr,
                                       paged_k_cache_ptr,
                                       paged_v_cache_ptr,
                                       output_ptr,
                                       paged_kv_indices_buffer_ptr,
                                       seq_len,
                                       qk_norm,
                                       rope,
                                       q_norm_weight_ptr,
                                       k_norm_weight_ptr,
                                       cos_ptr,
                                       sin_ptr,
                                       q_eps,
                                       k_eps);
}

template <typename T,
          int NUM_Q_PER_KV,
          int HEAD_DIM,
          int PAGE_SIZE,
          int MAX_SEQ_LEN,
          int KV_STRIDE>
void launch_paged_attention(void const *qkv_ptr,
                            void *paged_k_cache_ptr,
                            void *paged_v_cache_ptr,
                            void *output_ptr,
                            void const *paged_kv_indices_buffer_ptr,
                            size_t seq_len,
                            bool qk_norm,
                            bool rope,
                            void const *q_norm_weight_ptr,
                            void const *k_norm_weight_ptr,
                            void const *cos_ptr,
                            void const *sin_ptr,
                            float q_eps,
                            float k_eps) {
  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  size_t smem_size = 112640;

  hipFuncSetAttribute(reinterpret_cast<const void*>(paged_attention_wrapper<T),
                                               NUM_Q_PER_KV,
                                               HEAD_DIM,
                                               PAGE_SIZE,
                                               MAX_SEQ_LEN,
                                               KV_STRIDE>,
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       smem_size);

  paged_attention_wrapper<T,
                          NUM_Q_PER_KV,
                          HEAD_DIM,
                          PAGE_SIZE,
                          MAX_SEQ_LEN,
                          KV_STRIDE>
      <<<grid_dim, block_dim, smem_size>>>(qkv_ptr,
                                           paged_k_cache_ptr,
                                           paged_v_cache_ptr,
                                           output_ptr,
                                           paged_kv_indices_buffer_ptr,
                                           seq_len,
                                           qk_norm,
                                           rope,
                                           q_norm_weight_ptr,
                                           k_norm_weight_ptr,
                                           cos_ptr,
                                           sin_ptr,
                                           q_eps,
                                           k_eps);
}

void paged_attention(
    torch::Tensor qkv,
    torch::Tensor paged_k_cache,
    torch::Tensor paged_v_cache,
    torch::Tensor output,
    torch::Tensor paged_kv_indices_buffer,
    size_t seq_len,
    bool qk_norm,
    bool rope,
    torch::optional<torch::Tensor> q_norm_weight = torch::nullopt,
    torch::optional<torch::Tensor> k_norm_weight = torch::nullopt,
    torch::optional<torch::Tensor> cos = torch::nullopt,
    torch::optional<torch::Tensor> sin = torch::nullopt,
    float q_eps = 0.0f,
    float k_eps = 0.0f) {
  void const *qkv_ptr = qkv.data_ptr();
  void *paged_k_cache_ptr = paged_k_cache.data_ptr();
  void *paged_v_cache_ptr = paged_v_cache.data_ptr();
  void *output_ptr = output.data_ptr();
  void const *paged_kv_indices_buffer_ptr = paged_kv_indices_buffer.data_ptr();

  void const *q_norm_weight_ptr = qk_norm ? q_norm_weight->data_ptr() : nullptr;
  void const *k_norm_weight_ptr = qk_norm ? k_norm_weight->data_ptr() : nullptr;
  void const *cos_ptr = rope ? cos->data_ptr() : nullptr;
  void const *sin_ptr = rope ? sin->data_ptr() : nullptr;

  launch_paged_attention<bfloat16, 4, 128, 64, 512, 128>(
      qkv_ptr,
      paged_k_cache_ptr,
      paged_v_cache_ptr,
      output_ptr,
      paged_kv_indices_buffer_ptr,
      seq_len,
      qk_norm,
      rope,
      q_norm_weight_ptr,
      k_norm_weight_ptr,
      cos_ptr,
      sin_ptr,
      q_eps,
      k_eps);

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// RMSNorm Linear

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
__global__ void norm_linear_kernel_wrapper(void const *input_ptr,
                                           void const *norm_weight_ptr,
                                           void const *weight_ptr,
                                           float eps,
                                           void *output_ptr) {
  norm_linear_task_impl<T,
                        BATCH_SIZE,
                        OUTPUT_SIZE,
                        REDUCTION_SIZE,
                        OUTPUT_SIZE>(
      input_ptr, norm_weight_ptr, weight_ptr, eps, output_ptr);
}

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
void launch_norm_linear(void const *input_ptr,
                        void const *norm_weight_ptr,
                        void const *weight_ptr,
                        float eps,
                        void *output_ptr) {
  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  size_t smem_size = 1024 * 99;

  hipFuncSetAttribute(reinterpret_cast<const void*>(
      norm_linear_kernel_wrapper<T), BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>,
      hipFuncAttributeMaxDynamicSharedMemorySize,
      smem_size);

  norm_linear_kernel_wrapper<T, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>
      <<<grid_dim, block_dim, smem_size>>>(
          input_ptr, norm_weight_ptr, weight_ptr, eps, output_ptr);
}

void norm_linear(torch::Tensor input,
                 torch::Tensor norm_weight,
                 torch::Tensor weight,
                 float eps,
                 torch::Tensor output) {

  void const *input_ptr = input.data_ptr();
  void const *norm_weight_ptr = norm_weight.data_ptr();
  void const *weight_ptr = weight.data_ptr();
  void *output_ptr = output.data_ptr();

  switch (output.size(1)) {
    case 16:
      launch_norm_linear<bfloat16, 1, 16, 4096>(
          input_ptr, norm_weight_ptr, weight_ptr, eps, output_ptr);
      break;
    case 32:
      launch_norm_linear<bfloat16, 1, 32, 4096>(
          input_ptr, norm_weight_ptr, weight_ptr, eps, output_ptr);
      break;
    case 64:
      launch_norm_linear<bfloat16, 1, 64, 4096>(
          input_ptr, norm_weight_ptr, weight_ptr, eps, output_ptr);
      break;
    case 256:
      launch_norm_linear<bfloat16, 1, 256, 4096>(
          input_ptr, norm_weight_ptr, weight_ptr, eps, output_ptr);
      break;
    case 1600:
      launch_norm_linear<bfloat16, 1, 1600, 4096>(
          input_ptr, norm_weight_ptr, weight_ptr, eps, output_ptr);
      break;
    default:
      printf("Unsupported output size in test: %zu\n", output.size(1));
      break;
  }

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// Window RMSNorm Linear

template <typename T, int BATCH_SIZE, int WINDOW_SIZE, int HEAD_DIM>
__global__ void window_rms_norm_kernel_wrapper(void const *input_ptr,
                                          void const *weight_ptr,
                                          float eps,
                                          void *output_ptr,
                                          bool rotary_emd = false,
                                          T const *cos_ptr = nullptr,
                                          T const *sin_ptr = nullptr) {
  constexpr size_t q_num = BATCH_SIZE * WINDOW_SIZE;
  using Smem = kernel::smem_row<T, 3, 3, 3, q_num, 128, 128>;

  extern __shared__ char smem[];
  T* smem_ptr = reinterpret_cast<T*>(smem);
  Smem input_smem(smem_ptr);

  // TODO(Wenqin): q_num * HEAD_DIM is a conservative number.
  float* reduce_smem = reinterpret_cast<float*>(smem) + q_num * HEAD_DIM;

  T const* d_input = static_cast<T const*>(input_ptr);
  T* d_output = const_cast<T*>(static_cast<T const*>(output_ptr));

  kernel::dmem_row_const<T, q_num, 128, 128> input_dmem(d_input);
  kernel::dmem_row<T, q_num, 128, 128> output_dmem(d_output);

  for (int i = threadIdx.x; i < q_num * (HEAD_DIM / 8); i += NUM_THREADS) {
    int row = i / 16;
    int col = (i % 16) * 8;
    kernel::load_smem(input_smem(row, col), input_dmem(row, col));
  }
  kernel::cp_async_fence();
  kernel::cp_async_wait<0>();

  T const* norm_weight_ptr = static_cast<T const*>(weight_ptr);

  kernel::window_rms_norm<T, Smem, 1,
                        WINDOW_SIZE,
                        HEAD_DIM>(
      input_smem, norm_weight_ptr, reduce_smem, eps, rotary_emd, cos_ptr, sin_ptr);

  __syncthreads();

  for (int i = threadIdx.x; i < q_num * (HEAD_DIM / 8); i += NUM_THREADS) {
    // write back
    int row = i / 16;
    int col = (i % 16) * 8;
    for(int j = 0; j < 8; j ++) {
      int col_j = col + j;
      *output_dmem(row, col_j) = *input_smem(row, col_j);
    }
  }
}

#define WINDOW_RMSNORM_LINEAR_LAUNCHER(HEAD_DIM, WINDOW_SIZE)            \
  launch_window_rms_norm<bfloat16, 1, WINDOW_SIZE, HEAD_DIM>(         \
      input_ptr, weight_ptr, eps, output_ptr);

#define DISPATCH_WINDOW_RMSNORM_LINEAR_WINDOW_SIZE(HEAD_DIM)            \
  switch (window_size) {                                                   \
    case 1:                                                                \
      WINDOW_RMSNORM_LINEAR_LAUNCHER(HEAD_DIM, 1);                       \
      break;                                                               \
    case 2:                                                                \
      WINDOW_RMSNORM_LINEAR_LAUNCHER(HEAD_DIM, 2);                       \
      break;                                                               \
    case 3:                                                                \
      WINDOW_RMSNORM_LINEAR_LAUNCHER(HEAD_DIM, 3);                       \
      break;                                                               \
    case 4:                                                                \
      WINDOW_RMSNORM_LINEAR_LAUNCHER(HEAD_DIM, 4);                       \
      break;                                                               \
    default:                                                               \
      printf("Unsupported window size in test: %zu\n", window_size);       \
      break;                                                               \
  }

#define DISPATCH_WINDOW_RMSNORM_LINEAR_HEAD_DIM()                       \
  switch (head_dim) {                                                   \
    case 16:                                                               \
      DISPATCH_WINDOW_RMSNORM_LINEAR_WINDOW_SIZE(16);                       \
      break;                                                               \
    case 32:                                                               \
      DISPATCH_WINDOW_RMSNORM_LINEAR_WINDOW_SIZE(32);                       \
      break;                                                               \
    case 64:                                                               \
      DISPATCH_WINDOW_RMSNORM_LINEAR_WINDOW_SIZE(64);                       \
      break;                                                               \
    case 128:                                                               \
      DISPATCH_WINDOW_RMSNORM_LINEAR_WINDOW_SIZE(128);                       \
      break;                                                               \
    case 256:                                                              \
      DISPATCH_WINDOW_RMSNORM_LINEAR_WINDOW_SIZE(256);                      \
      break;                                                               \
    case 1600:                                                             \
      DISPATCH_WINDOW_RMSNORM_LINEAR_WINDOW_SIZE(1600);                     \
      break;                                                               \
    default:                                                               \
      printf("Unsupported head dim in test: %zu\n", head_dim);       \
      break;                                                               \
  }

#define WINDOW_RMSNORM_LINEAR() DISPATCH_WINDOW_RMSNORM_LINEAR_HEAD_DIM()

template <typename T, int BATCH_SIZE, int WINDOW_SIZE, int HEAD_DIM>
void launch_window_rms_norm(void const *input_ptr,
                        void const *weight_ptr,
                        float eps,
                        void *output_ptr) {
  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  size_t smem_size = 1024 * 99;

  hipFuncSetAttribute(reinterpret_cast<const void*>(
      window_rms_norm_kernel_wrapper<T), BATCH_SIZE, WINDOW_SIZE, HEAD_DIM>,
      hipFuncAttributeMaxDynamicSharedMemorySize,
      smem_size);

  window_rms_norm_kernel_wrapper<T, BATCH_SIZE, WINDOW_SIZE, HEAD_DIM>
      <<<grid_dim, block_dim, smem_size>>>(
          input_ptr, weight_ptr, eps, output_ptr);
}

void window_rms_norm(torch::Tensor input, // shape [batch, window_size, head_dim]
                 torch::Tensor weight,
                 float eps,
                 torch::Tensor output) {
  void const *input_ptr = input.data_ptr();
  void const *weight_ptr = weight.data_ptr();
  void *output_ptr = output.data_ptr();
  size_t head_dim = output.size(2);
  size_t window_size = output.size(1);

  DISPATCH_WINDOW_RMSNORM_LINEAR_HEAD_DIM();

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA window_norm_linear kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// SiLU MUL Linear

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
__global__ void silu_mul_linear_kernel_wrapper(void const *input_ptr,
                                               void const *weight_ptr,
                                               void const *bias_ptr,
                                               void *output_ptr) {
  silu_mul_linear_task_impl<T, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>(
      input_ptr, weight_ptr, bias_ptr, output_ptr);
}

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
void launch_silu_mul_linear(void const *input_ptr,
                            void const *weight_ptr,
                            void const *bias_ptr,
                            void *output_ptr) {
  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  size_t smem_size = 112640;

  hipFuncSetAttribute(reinterpret_cast<const void*>(silu_mul_linear_kernel_wrapper<T),
                                                      BATCH_SIZE,
                                                      OUTPUT_SIZE,
                                                      REDUCTION_SIZE>,
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       smem_size);

  silu_mul_linear_kernel_wrapper<T, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>
      <<<grid_dim, block_dim, smem_size>>>(
          input_ptr, weight_ptr, bias_ptr, output_ptr);
}

void silu_mul_linear(torch::Tensor input,
                     torch::Tensor weight,
                     torch::Tensor bias,
                     torch::Tensor output) {

  void const *input_ptr = input.data_ptr();
  void const *weight_ptr = weight.data_ptr();
  void const *bias_ptr = bias.data_ptr();
  void *output_ptr = output.data_ptr();

  switch (output.size(1)) {
    case 16:
      launch_silu_mul_linear<bfloat16, 1, 16, 12288>(
          input_ptr, weight_ptr, bias_ptr, output_ptr);
      break;
    case 32:
      launch_silu_mul_linear<bfloat16, 1, 32, 12288>(
          input_ptr, weight_ptr, bias_ptr, output_ptr);
      break;
    case 64:
      launch_silu_mul_linear<bfloat16, 1, 64, 12288>(
          input_ptr, weight_ptr, bias_ptr, output_ptr);
      break;
    default:
      printf("Unsupported output size in test: %zu\n", output.size(1));
      break;
  }

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// Linear

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
__global__ void linear_kernel_wrapper(void const *input_ptr,
                                      void const *weight_ptr,
                                      void const *residual_ptr,
                                      void *output_ptr) {
  linear_kernel<T, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>(
      input_ptr, weight_ptr, residual_ptr, output_ptr);
}

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
void launch_linear(void const *input_ptr,
                   void const *weight_ptr,
                   void const *residual_ptr,
                   void *output_ptr) {
  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  size_t smem_size = 112640;

  hipFuncSetAttribute(reinterpret_cast<const void*>(
      linear_kernel_wrapper<T), BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>,
      hipFuncAttributeMaxDynamicSharedMemorySize,
      smem_size);

  linear_kernel_wrapper<T, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>
      <<<grid_dim, block_dim, smem_size>>>(
          input_ptr, weight_ptr, residual_ptr, output_ptr);
}

void linear(torch::Tensor input,
            torch::Tensor weight,
            torch::Tensor residual,
            torch::Tensor output) {

  void const *input_ptr = input.data_ptr();
  void const *weight_ptr = weight.data_ptr();
  void const *residual_ptr = residual.data_ptr();
  void *output_ptr = output.data_ptr();

  switch (output.size(1)) {
    case 16:
      launch_linear<bfloat16, 1, 16, 4096>(
          input_ptr, weight_ptr, residual_ptr, output_ptr);
      break;
    case 32:
      launch_linear<bfloat16, 1, 32, 4096>(
          input_ptr, weight_ptr, residual_ptr, output_ptr);
      break;
    case 64:
      launch_linear<bfloat16, 1, 64, 4096>(
          input_ptr, weight_ptr, residual_ptr, output_ptr);
      break;
    default:
      printf("Unsupported output size in test: %zu\n", output.size(1));
      break;
  }

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// Argmax

// template <typename T>
// __global__ void argmax_kernel_wrapper(void const *input_ptr, void
// *output_ptr) {
//   argmax_kernel<T, 1, 32768>(input_ptr, output_ptr);
// }

// template <typename T>
// void launch_argmax(void const *input_ptr, void *output_ptr) {
//   dim3 grid_dim(1, 1, 1);
//   dim3 block_dim(128, 1, 1);
//   size_t smem_size = 36666;

//   hipFuncSetAttribute(reinterpret_cast<const void*>(argmax_kernel_wrapper<T>),
//                        hipFuncAttributeMaxDynamicSharedMemorySize,
//                        smem_size);

//   argmax_kernel_wrapper<T>
//       <<<grid_dim, block_dim, smem_size>>>(input_ptr, output_ptr);
// }

// void argmax(torch::Tensor input, torch::Tensor output) {

//   void const *input_ptr = input.data_ptr();
//   void *output_ptr = output.data_ptr();

//   launch_argmax<bfloat16>(input_ptr, output_ptr);

//   hipError_t err = hipDeviceSynchronize();
//   if (err != hipSuccess) {
//     printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
//   }
// }

// pybind11 bindings

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("linear", &linear, "Linear kernel");
  // m.def("argmax", &argmax, "argmax kernel");
  m.def("norm_linear", &norm_linear, "RMSNorm Linear kernel");
  m.def("silu_mul_linear", &silu_mul_linear, "SILU MUL Linear kernel");
  m.def("single_batch_decoding",
        &single_batch_decoding,
        py::arg("qkv"),
        py::arg("k_cache"),
        py::arg("v_cache"),
        py::arg("output"),
        py::arg("seq_len"),
        py::arg("qk_norm"),
        py::arg("rotary_embed"),
        py::arg("qnorm_weight") = py::none(),
        py::arg("knorm_weight") = py::none(),
        py::arg("cos") = py::none(),
        py::arg("sin") = py::none(),
        py::arg("q_eps") = 0.0f,
        py::arg("k_eps") = 0.0f);
  m.def("paged_attention", &paged_attention, "Paged Attention");
  m.def("window_rms_norm", &window_rms_norm, "Window RMSNorm");
}